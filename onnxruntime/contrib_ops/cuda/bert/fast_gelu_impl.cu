#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on gelu plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_call.h"
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include "fast_gelu_impl.h"

using namespace onnxruntime::cuda;

namespace onnxruntime {
namespace contrib {
namespace cuda {

// constants for approximating the normal cdf
constexpr float A = 0.5;

constexpr float B = 0.7978845608028654;  // sqrt(2.0/M_PI)

constexpr float C = 0.035677408136300125;  // 0.044715 * sqrt(2.0/M_PI)

__device__ inline float tanh(const float& x) {
  return tanhf(x);
}

__device__ inline half tanh(const half& x) {
  const float tmp = tanhf(__half2float(x));
  return __float2half(tmp);
}

__device__ inline half2 tanh(const half2& x) {
  // at the moment, there is no half2 tanh builtin
  float2 tmp = (__half22float2(x));
  tmp.x = tanhf(tmp.x);
  tmp.y = tanhf(tmp.y);
  return __float22half2_rn(tmp);
}

template <typename T, unsigned TPB>
__global__ void geluKernel(const T a, const T b, const T c, int input_length, int bias_length, const T* input, const T* bias, T* output) {
  const int idx = blockIdx.x * TPB + threadIdx.x;

  if (idx < input_length) {
    const T x = input[idx];
    const T in = (bias == nullptr) ? x : (x + bias[idx % bias_length]);
    const T cdf = a + a * tanh(in * (c * in * in + b));
    output[idx] = in * cdf;
  }
}

template<>
bool computeGelu<float>(hipStream_t stream, int input_length, int bias_length, const float* input, const float* bias, float* output) {
  constexpr int blockSize = 256;
  const int gridSize = (input_length + blockSize - 1) / blockSize;
  geluKernel<float, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);

  return CUDA_CALL(hipPeekAtLastError());
}

template<>
bool computeGelu<half>(hipStream_t stream, int input_length, int bias_length, const half* input, const half* bias, half* output) {
  const int blockSize = 256;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
  if (0 == (bias_length & 1)) {
    const int n = input_length / 2;
    const int gridSize = (n + blockSize - 1) / blockSize;
    const half2 A2 = __floats2half2_rn(A, A);
    const half2 B2 = __floats2half2_rn(B, B);
    const half2 C2 = __floats2half2_rn(C, C);
    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    geluKernel<half2, blockSize><<<gridSize, blockSize, 0, stream>>>(A2, B2, C2, n, bias_length / 2, input2, bias2, output2);
  } else 
#endif
  {
    const int gridSize = (input_length + blockSize - 1) / blockSize;
    geluKernel<half, blockSize><<<gridSize, blockSize, 0, stream>>>(A, B, C, input_length, bias_length, input, bias, output);
  }

  return CUDA_CALL(hipPeekAtLastError());
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
