#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on skipLayerNorm plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "contrib_ops/cuda/bert/layer_norm.cuh"
#include "contrib_ops/cuda/bert/skip_layer_norm_impl.h"
#include <hip/hip_fp16.h>

namespace onnxruntime {
namespace contrib {
namespace cuda {

constexpr float one = 1.0;

template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernelSmall(
    const int ld, const T* input, const T* skip, const T* beta, const T* gamma, const T* bias, 
    const T epsilon, T* output) {
  const T reverse_ld = T(1.f / ld);
  const int offset = blockIdx.x * ld;

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;

  if (threadIdx.x < ld) {
    val = (bias == nullptr) ? input[idx] + skip[idx] : input[idx] + skip[idx] + bias[threadIdx.x];
    const T rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
  }

  LayerNormSmall<T, TPB>(val, thread_data, ld, idx, beta, gamma, epsilon, output);
}

template <typename T, unsigned TPB> // TODO: T is redundant here!
__global__ void SkipLayerNormKernelSmall2(
    const int ld, const half2* input, const half2* skip, const half2* beta,
    const half2* gamma, const half2* bias, const half2 epsilon, half2* output) {
  // const half2 reverse_ld = T(1.f / ld);
  //const half2 reverse_ld = h2rcp(__float2half2_rn(float(ld))); // TODO
  
  // workaround for a llvm bug: https://github.com/intel/llvm/issues/5153
  const half2 one2 = __float2half2_rn(one);
  const half2 ld2 = __float2half2_rn(float(ld));
  const half2 reverse_ld = one2 / ld2;
  
  /*
  const half2 ld2 = __float2half2_rn(float(ld));
  const half2 reverse_ld = h2rcp(ld2);
  */
  const int offset = blockIdx.x * ld; // shall I refactor this offset

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  // hipcub::KeyValuePair<half2, half2> thread_data(0, 0); // TODO: How to initialize a half2 pair
  hipcub::KeyValuePair<half2, half2> thread_data(__float2half2_rn(float(0.0)), __float2half2_rn(float(0.0))); // TODO: How to initialize a half2 pair
  const int idx = offset + threadIdx.x;
  half2 val = __float2half2_rn(float(0.0));

  if (threadIdx.x < ld) {
    val = (bias == nullptr) ? input[idx] + skip[idx] : input[idx] + skip[idx] + bias[threadIdx.x];
    const half2 rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<half2, half2>(rldval, rldval * val));
  }

  LayerNormSmall<half2, TPB>(val, thread_data, ld, idx, beta, gamma, epsilon, output);
}

template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernel(
    const int ld, const T* input, const T* skip, const T* beta, const T* gamma, const T* bias, 
    const T epsilon, T* output) {
  const T reverse_ld = T(1.f / ld);
  const int offset = blockIdx.x * ld;

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> thread_data(0, 0);

  for (int i = threadIdx.x; i < ld; i += TPB) {
    const int idx = offset + i;
    const T val = (bias == nullptr) ? input[idx] + skip[idx] : input[idx] + skip[idx] + bias[i];
    const T rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
    output[idx] = val;
  }

  LayerNorm<T, TPB>(thread_data, ld, offset, beta, gamma, epsilon, output);
}

template <typename T, unsigned TPB> // TODO: T is redundant here!
__global__ void SkipLayerNormKernel2(
    const int ld, const half2* input, const half2* skip, const half2* beta,
    const half2* gamma, const half2* bias, const half2 epsilon, half2* output) {
  // const half2 reverse_ld = T(1.f / ld);
  //const half2 reverse_ld = h2rcp(__float2half2_rn(float(ld))); // TODO
  const half2 one2 = __float2half2_rn(one);
  const half2 ld2 = __float2half2_rn(float(ld));
  const half2 reverse_ld = one2 / ld2; 
  const int offset = blockIdx.x * ld; // shall I refactor this offset

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  // hipcub::KeyValuePair<half2, half2> thread_data(0, 0); // TODO: How to initialize a half2 pair
  hipcub::KeyValuePair<half2, half2> thread_data(__float2half2_rn(float(0.0)), __float2half2_rn(float(0.0))); // TODO: How to initialize a half2 pair
  const int idx = offset + threadIdx.x;
  half2 val = __float2half2_rn(float(0.0)); // TODO: Can I initialize half2 like this?

  for (int i = threadIdx.x; i < ld; i += TPB) {
    // val = (bias == nullptr) ? input[idx] + skip[idx] : input[idx] + skip[idx] + bias[threadIdx.x];
    // const half2 rldval = reverse_ld * val;
    // thread_data = pair_sum(thread_data, hipcub::KeyValuePair<half2, half2>(rldval, rldval * val));
    const int idx = offset + i;
    const half2 val = (bias == nullptr) ? input[idx] + skip[idx] : input[idx] + skip[idx] + bias[i];
    const half2 rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<half2, half2>(rldval, rldval * val));
    output[idx] = val;
  }

  LayerNormSmall<half2, TPB>(val, thread_data, ld, idx, beta, gamma, epsilon, output);
}

template <typename T>
bool ComputeSkipLayerNorm(
    const hipDeviceProp_t& prop, hipStream_t stream, const int ld, const int n, const T* input,
    const T* skip, const T* beta, const T* gamma, const T* bias, const T epsilon, T* output, bool use_half2) {
  // this must be true because n is the total size of the tensor
  assert(n % ld == 0);
  if (use_half2 && 0 == (n & 1) && prop.major >= 7) {
    const int n2 = n / 2;
    const int grid_size = n2 / ld;

    const half2* input2 = reinterpret_cast<const half2*>(input);
    const half2* skip2 = reinterpret_cast<const half2*>(skip);
    const half2* beta2 = reinterpret_cast<const half2*>(beta);
    const half2* gamma2 = reinterpret_cast<const half2*>(gamma);
    const half2* bias2 = reinterpret_cast<const half2*>(bias);
    half2* output2 = reinterpret_cast<half2*>(output);
    const half2 epsilon2 = __float2half2_rn(epsilon);

    if (ld <= 32) {
      constexpr int block_size = 32;
      SkipLayerNormKernelSmall2<half2, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input2, skip2, beta2, gamma2, bias2, epsilon2, output2);
      // TODO: ld / 2 ???
    } else if (ld <= 128) {
      constexpr int block_size = 128;
      SkipLayerNormKernelSmall2<half2, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input2, skip2, beta2, gamma2, bias2, epsilon2, output2);
    } else if (ld == 384) {
      constexpr int block_size = 384;
      SkipLayerNormKernelSmall2<half2, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input2, skip2, beta2, gamma2, bias2, epsilon2, output2);
    } else {
      // TODO: check if half2 also works for this function or not
      constexpr int block_size = 256;
      SkipLayerNormKernel2<half2, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input2, skip2, beta2, gamma2, bias2, epsilon2, output2);
    }
  } else {
    const int grid_size = n / ld;
    if (ld <= 32) {
      constexpr int block_size = 32;
      SkipLayerNormKernelSmall<T, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
    } else if (ld <= 128) {
      constexpr int block_size = 128;
      SkipLayerNormKernelSmall<T, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
    } else if (ld == 384) {
      constexpr int block_size = 384;
      SkipLayerNormKernelSmall<T, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
    } else {
      constexpr int block_size = 256;
      SkipLayerNormKernel<T, block_size>
          <<<grid_size, block_size, 0, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
    }
  }
  return CUDA_CALL(hipPeekAtLastError());
}

bool LaunchSkipLayerNormKernel(
    const hipDeviceProp_t& prop,
    hipStream_t stream,
    void* output,
    const void* input,
    const void* skip,
    const void* gamma,
    const void* beta,
    const void* bias,    
    float epsilon,
    int hidden_size,
    int element_count,
    size_t element_size,
    bool use_half2) {
  if (element_size == 2) {
    return ComputeSkipLayerNorm(
        prop,
        stream,
        hidden_size,
        element_count,
        reinterpret_cast<const half*>(input),
        reinterpret_cast<const half*>(skip),
        reinterpret_cast<const half*>(beta),
        reinterpret_cast<const half*>(gamma),
        reinterpret_cast<const half*>(bias),
        __float2half_rn(epsilon),
        reinterpret_cast<half*>(output),
        use_half2);
  } else {
    return ComputeSkipLayerNorm(
        prop,
        stream,
        hidden_size,
        element_count,
        reinterpret_cast<const float*>(input),
        reinterpret_cast<const float*>(skip),
        reinterpret_cast<const float*>(beta),
        reinterpret_cast<const float*>(gamma),
        reinterpret_cast<const float*>(bias),
        epsilon,
        reinterpret_cast<float*>(output),
        false);
  }
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
