#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cuda_common.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "cub/util_type.cuh"
#include <hipcub/hipcub.hpp>
#include <cub/device/device_segmented_radix_sort.cuh>
#include "contrib_ops/cuda/bert/utils.cuh"
#include "contrib_ops/cuda/transformers/generation_cuda_impl.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {
__global__ void InitKernel(float* beam_scores,
                           int num_beams,
                           int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int beam_index = index % num_beams;
    beam_scores[index] = beam_index > 0 ? static_cast<float>(-1e9) : 0.0f;
  }
}

void LaunchInitKernel(
    float* beam_scores,
    int batch_size,
    int num_beams,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  InitKernel<<<gridSize, blockSize, 0, stream>>>(beam_scores, num_beams, total_elements);
}

__global__ void NextTokenKernel(const int64_t* next_token_indices,
                                int32_t* next_indices,
                                int32_t* next_tokens,
                                int vocab_size,
                                int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    next_indices[index] = next_token_indices[index] / vocab_size;
    next_tokens[index] = next_token_indices[index] % vocab_size;
  }
}

void LaunchNextTokenKernel(const int64_t* next_token_indices,
                           int32_t* next_indices,
                           int32_t* next_tokens,
                           int batch_size,
                           int top_k,
                           int vocab_size,
                           hipStream_t stream) {
  int total_elements = batch_size * top_k;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  NextTokenKernel<<<gridSize, blockSize, 0, stream>>>(next_token_indices,
                                                      next_indices,
                                                      next_tokens,
                                                      vocab_size,
                                                      total_elements);
}

template <typename T>
__global__ void LogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    const int* presence_mask,
    float presence_penalty,
    float temperature,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int total_elements,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int batch_beam_index = index / padded_vocab_size;
    int word_id = index % padded_vocab_size;

    if (word_id >= vocab_size) {
      // Set any value within the padding region to the lowest value so that it isn't picked
      next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
    } else {
      // RepetitionPenaltyLogitsProcessor
      if (repetition_penalty != 1.0f) {
        const int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = 0; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {
            found = true;
            break;
          }
        }
        if (found) {
          float score = (float)next_token_scores[index];
          next_token_scores[index] = (T)(score < 0 ? score * repetition_penalty : score / repetition_penalty);
        }
      }

      // NoRepeatNGramLogitsProcessor
      if (no_repeat_ngram_size > 0 && current_sequence_length >= no_repeat_ngram_size) {
        const int32_t* current_sequence = sequences + batch_beam_index * max_sequence_length;
        bool found = false;
        for (int i = no_repeat_ngram_size - 1; i < current_sequence_length; i++) {
          if (current_sequence[i] == word_id) {  // last token of n-gram matched
            found = true;
            for (int j = 0; j < no_repeat_ngram_size - 1; j++) {  // match the remaining N-1 tokens
              if (current_sequence[i - j - 1] != current_sequence[current_sequence_length - 1 - j]) {
                found = false;
                break;
              }
            }
            if (found) {
              break;
            }
          }
        }

        if (found) {
          next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
          return;
        }
      }

      // VocabMaskLogitsProcessor
      if (vocab_mask != nullptr && vocab_mask[word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // PrefixVocabMaskLogitsProcessor
      int batch_id = batch_beam_index / num_beams;
      if (prefix_vocab_mask != nullptr && prefix_vocab_mask[batch_id * vocab_size + word_id] == 0) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
        return;
      }

      // MinLengthLogitsProcessor
      if (word_id == demote_token_id) {
        next_token_scores[index] = hipcub::FpLimits<T>::Lowest();
      }

      // PresencePenaltyLogitsProcessor
      if (presence_mask != nullptr && presence_mask[index] == 1) {
        float score = (float)next_token_scores[index] - presence_penalty;
        next_token_scores[index] = (T)score;
      }

      // TemperatureLogitsProcessor
      if (temperature != 1.0f) {
        float score = (float)(next_token_scores[index]);
        next_token_scores[index] = (T)(score / temperature);
      }
    }
  }
}

template <typename T>
void LaunchLogitsProcessKernel(
    T* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams * padded_vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  LogitsProcessKernel<T><<<gridSize, blockSize, 0, stream>>>(
      next_token_scores,
      vocab_mask,
      prefix_vocab_mask,
      presence_mask,
      presence_penalty,
      temperature,
      num_beams,
      vocab_size,
      padded_vocab_size,
      total_elements,
      demote_token_id,
      sequences,
      max_sequence_length,
      current_sequence_length,
      repetition_penalty,
      no_repeat_ngram_size);
}

// Instantiation
template void LaunchLogitsProcessKernel(
    float* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

template void LaunchLogitsProcessKernel(
    half* next_token_scores,
    const int* vocab_mask,
    const int* prefix_vocab_mask,
    int* presence_mask,
    float presence_penalty,
    float temperature,
    int batch_size,
    int num_beams,
    int vocab_size,
    int padded_vocab_size,
    int demote_token_id,
    const int32_t* sequences,
    int max_sequence_length,
    int current_sequence_length,
    float repetition_penalty,
    int no_repeat_ngram_size,
    hipStream_t stream);

__global__ void InitializeBeamHypotheses(gsl::span<BeamHypotheses> beam_hyps, float length_penalty, gsl::span<HypothesisScore> beams, int num_beams) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= beam_hyps.size())
    return;

  BeamHypotheses& beam_hyp = beam_hyps[index];
  beam_hyp.beams_ = beams.subspan(index * num_beams, num_beams);
  beam_hyp.beams_used_ = 0;
  beam_hyp.length_penalty_ = length_penalty;
  beam_hyp.done_ = false;
}

void LaunchInitializeBeamHypotheses(gsl::span<BeamHypotheses> beam_hyps, float length_penalty, gsl::span<HypothesisScore> beams, int num_beams, hipStream_t stream) {
  int block_size = (beam_hyps.size() + 31) & ~31;  // Round up to nearest multiple of 32
  int grid_size = 1;
  if (block_size > 256) {
    grid_size = (block_size + 255) / 256;
    block_size = 256;
  }

  InitializeBeamHypotheses<<<grid_size, block_size, 0, stream>>>(beam_hyps, length_penalty, beams, num_beams);
}

__device__ void BeamHypotheses::Add(const gsl::span<const int32_t> hypothesis, float sum_logprobs) {
  auto length = hypothesis.size();
  float score = sum_logprobs / pow(static_cast<float>(length), length_penalty_);

  size_t index = beams_used_;
  // If the array is full, don't add unless it's better than the worst element
  if (index == beams_.size()) {
    if (score <= beams_[--index].score)
      return;
  } else
    beams_used_++;

  // Rotate existing elements over while the new element scores higher
  for (; index > 0 && score > beams_[index - 1].score; index--)
    beams_[index] = beams_[index - 1];

  beams_[index] = HypothesisScore{hypothesis, score};
}

__device__ bool BeamHypotheses::CanImprove(float best_sum_logprobs, int current_length) const {
  float current_score = best_sum_logprobs / pow(static_cast<float>(current_length), length_penalty_);
  return beams_.back().score < current_score;
}

__device__ void BeamHypotheses::Output(
    int top_k,
    int max_length,
    gsl::span<int32_t> sequences,       // buffer filled with pad token ID, shape (num_return_sequences, max_length)
    gsl::span<float> sequences_scores)  // buffer of shape (num_return_sequences) or empty
{
  // Copy the top_k beams into the sequences
  for (int index = 0; index < top_k; index++) {
    auto& item = beams_[index];
    gsl::span<int32_t> target = sequences.subspan(static_cast<gsl::index>(index) * max_length, max_length);

    // Note that word_ids might be less than max_length.
    // Since the sequences has been filled with pad token ID, so padding is not needed here.
    for (int i = 0; i < target.size(); i++)
      target[i] = item.hypothesis[i];

    if (!sequences_scores.empty())
      sequences_scores[index] = item.score;
  }
}

__global__ void BeamSearchScorer_Process(BeamScorerState& state,
                                         gsl::span<const int32_t> sequences_buffer,
                                         gsl::span<int32_t> next_sequences,
                                         int sequence_length,
                                         gsl::span<BeamHypotheses> beam_hyps_,
                                         gsl::span<float> next_beam_scores_,
                                         gsl::span<int32_t> next_beam_tokens_,
                                         gsl::span<int32_t> next_beam_indices_,
                                         gsl::span<int32_t> hypothesis_buffer_,
                                         gsl::span<const float> next_scores,
                                         gsl::span<const int32_t> next_tokens,
                                         gsl::span<const int32_t> next_indices) {
  // Sequences shape is (batch_size * num_beams, total_sequence_length)
  // It contains word ID of whole sequence generated so far.
  // It is different from subgraph input_ids, which only need one word when past state is not empty.

  int batch = blockIdx.x * blockDim.x + threadIdx.x;
  if (batch >= state.batch_size_)
    return;

  int batch_start = batch * state.num_beams_;

  while (true) {  // Use a while loop so 'break' is equivalent to a goto outside of this scope
    cuda::BeamHypotheses& beam_hyp = beam_hyps_[batch];
    if (beam_hyp.done_) {
      // Pad the batch.
      for (size_t j = 0; j < state.num_beams_; j++) {
        next_beam_scores_[batch_start + j] = 0.0f;
        next_beam_tokens_[batch_start + j] = state.pad_token_id_;
        next_beam_indices_[batch_start + j] = 0;
      }
      break;
    }

    // Next tokens for this sentence.
    size_t beam_idx = 0;
    size_t top_k = 2 * state.num_beams_;
    for (size_t j = 0; j < top_k; j++) {
      int32_t next_token = next_tokens[batch * top_k + j];
      float next_score = next_scores[batch * top_k + j];
      int32_t next_index = next_indices[batch * top_k + j];

      int batch_beam_idx = batch_start + next_index;
      // Add to generated hypotheses if end of sentence.
      if ((state.eos_token_id_ >= 0) && (next_token == state.eos_token_id_)) {
        bool is_beam_token_worse_than_top_num_beams = (j >= state.num_beams_);
        if (is_beam_token_worse_than_top_num_beams) {
          continue;
        }

        // Clone the sequence and append to buffer.
        gsl::span<const int32_t> src = sequences_buffer.subspan(batch_beam_idx * state.max_length_, sequence_length);
        auto clone = hypothesis_buffer_.subspan(atomicAdd(&state.hypothesis_buffer_used_, sequence_length), sequence_length);

        for (unsigned i = 0; i < src.size(); i++)
          clone[i] = src[i];
        beam_hyp.Add(gsl::span<const int32_t>(clone.data(), clone.size()), next_score);
      } else {
        // Add next predicted token since it is not eos_token.
        next_beam_scores_[batch_start + beam_idx] = next_score;
        next_beam_tokens_[batch_start + beam_idx] = next_token;
        next_beam_indices_[batch_start + beam_idx] = batch_beam_idx;
        ++beam_idx;
      }

      // Once the beam for next step is full, don't add more tokens to it.
      if (beam_idx == state.num_beams_)
        break;
    }

    //  Check if we are done so that we can save a pad step if all(done)
    if (beam_hyp.beams_used_ < state.num_beams_)
      break;

    if (!state.early_stopping_) {
      gsl::span<const float> topk_scores = next_scores.subspan(batch_start, top_k);
      const auto best_sum_logprobs = std::max_element(topk_scores.begin(), topk_scores.end());
      if (beam_hyp.CanImprove(*best_sum_logprobs, sequence_length))
        break;
    }

    beam_hyp.done_ = true;
    state.not_done_count_--;
    break;
  }

  // AppendNextTokenToSequences
  for (int beam_idx = 0; beam_idx < state.num_beams_; beam_idx++) {
    int beam_index = next_beam_indices_[batch_start + beam_idx];
    const int32_t* source = &sequences_buffer[beam_index * state.max_length_];
    int32_t* target = &next_sequences[(batch_start + beam_idx) * state.max_length_];
    for (int i = 0; i < sequence_length; i++)
      target[i] = source[i];

    // Append next token to each beam.
    target[sequence_length] = next_beam_tokens_[batch_start + beam_idx];
  }
}

void LaunchBeamSearchScorer_Process(int batch_size,
                                    BeamScorerState& state,
                                    gsl::span<const int32_t> sequences,
                                    gsl::span<int32_t> next_sequences,
                                    int sequence_length,
                                    gsl::span<BeamHypotheses> beam_hyps,
                                    gsl::span<float> next_beam_scores,
                                    gsl::span<int32_t> next_beam_tokens,
                                    gsl::span<int32_t> next_beam_indices,
                                    gsl::span<int32_t> hypothesis_buffer,
                                    gsl::span<const float> next_scores,
                                    gsl::span<const int32_t> next_tokens,
                                    gsl::span<const int32_t> next_indices,
                                    hipStream_t stream) {
  BeamSearchScorer_Process<<<1, batch_size, 0, stream>>>(state,
                                                         sequences,
                                                         next_sequences,
                                                         sequence_length,
                                                         beam_hyps,
                                                         next_beam_scores,
                                                         next_beam_tokens,
                                                         next_beam_indices,
                                                         hypothesis_buffer,
                                                         next_scores,
                                                         next_tokens,
                                                         next_indices);
}

__global__ void BeamSearchScorer_Finalize(BeamScorerState& state,
                                          gsl::span<const int32_t> sequences_buffer,
                                          int sequence_length,
                                          gsl::span<BeamHypotheses> beam_hyps_,
                                          gsl::span<const float> final_beam_scores,
                                          gsl::span<int32_t> output,
                                          gsl::span<float> sequence_scores) {
  // Finalize all open beam hypotheses and add to generated hypotheses.
  for (size_t batch_index = 0; batch_index < state.batch_size_; batch_index++) {
    cuda::BeamHypotheses& beam_hyp = beam_hyps_[batch_index];
    if (beam_hyp.done_) {
      continue;
    }

    for (size_t beam_index = 0; beam_index < state.num_beams_; beam_index++) {
      size_t batch_beam_index = batch_index * state.num_beams_ + beam_index;
      float final_score = final_beam_scores[batch_beam_index];
      auto final_tokens = sequences_buffer.subspan(batch_beam_index * state.max_length_, sequence_length);
      //      auto final_tokens = sequences.GetSequence(batch_beam_index);
      beam_hyp.Add(final_tokens, final_score);
    }
  }

  // Fill output sequences with pad token ID so that we do not need append it later.
  for (size_t i = 0; i < output.size(); i++)
    output[i] = state.pad_token_id_;

  // Select the best hypotheses according to number of sequences to return.
  for (size_t batch_index = 0; batch_index < state.batch_size_; batch_index++) {
    cuda::BeamHypotheses& beam_hyp = beam_hyps_[batch_index];

    auto batch_output = output.subspan(batch_index * state.num_return_sequences_ * state.max_length_,
                                       state.num_return_sequences_ * state.max_length_);
    beam_hyp.Output(
        state.num_return_sequences_,
        state.max_length_,
        batch_output,
        sequence_scores.empty() ? sequence_scores : sequence_scores.subspan(batch_index * state.num_return_sequences_, state.num_return_sequences_));
  }
}

void LaunchBeamSearchScorer_Finalize(BeamScorerState& state,
                                     gsl::span<const int32_t> sequences,
                                     int sequence_length,
                                     gsl::span<BeamHypotheses> beam_hyps,
                                     gsl::span<const float> final_beam_scores,
                                     gsl::span<int32_t> output,
                                     gsl::span<float> sequence_scores,
                                     hipStream_t stream) {
  BeamSearchScorer_Finalize<<<1, 1, 0, stream>>>(state,
                                                 sequences,
                                                 sequence_length,
                                                 beam_hyps,
                                                 final_beam_scores,
                                                 output,
                                                 sequence_scores);
}

__global__ void AddProbsKernel(float* log_probs,
                               float* cum_log_probs,
                               const int vocab_size,
                               const int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_beam_index = index / vocab_size;

  if (index < total_elements)
    log_probs[index] += cum_log_probs[batch_beam_index];
}

template <typename T>
void LaunchAddProbsKernel(T* log_probs,
                          T* cum_log_probs,
                          const int batch_size,
                          const int num_beams,
                          const int vocab_size,
                          hipStream_t stream) {
  int total_elements = batch_size * num_beams * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  AddProbsKernel<<<gridSize, blockSize, 0, stream>>>(log_probs, cum_log_probs, vocab_size, total_elements);
}

template void LaunchAddProbsKernel(
    float* log_probs,
    float* cum_log_probs,
    const int batch_size,
    const int num_beams,
    const int vocab_size,
    hipStream_t stream);

template <typename T>
__global__ void UpdateGptInputsKernel(const T* old_mask_data,
                                      T* mask_data,
                                      int32_t* next_positions,
                                      int batch_beam_size,
                                      int current_length) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < batch_beam_size * current_length) {
    // Update attention mask.
    int i = index / current_length;
    int j = index % current_length;
    mask_data[index] = (j < current_length - 1) ? old_mask_data[i * (current_length - 1) + j] : static_cast<T>(1);

    if (next_positions != nullptr) {
      // Update sequence length (or next positions).
      if (index < batch_beam_size) {
        next_positions[index]++;
      }
    }
  }
}

void LaunchUpdateGptKernel(const int32_t* old_mask_data,
                           int32_t* mask_data,
                           int32_t* next_positions,
                           int batch_beam_size,
                           int current_length,
                           hipStream_t stream) {
  assert(current_length > 0);
  int total_elements = batch_beam_size * current_length;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  UpdateGptInputsKernel<int32_t><<<gridSize, blockSize, 0, stream>>>(
      old_mask_data, mask_data, next_positions, batch_beam_size, current_length);
}

template <typename T>
void GetTempStorageSize(const T* d_keys_in,
                        const int* d_values_in,
                        int* d_offsets,
                        int num_items,
                        int num_segments,
                        hipStream_t stream,
                        bool is_descending,
                        size_t& temp_storage_bytes) {
  if (is_descending) {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
                                                                       temp_storage_bytes,
                                                                       d_keys_in,
                                                                       (T*)nullptr,
                                                                       d_values_in,
                                                                       (int*)nullptr,
                                                                       num_items,
                                                                       num_segments,
                                                                       d_offsets,
                                                                       d_offsets + 1,
                                                                       0,
                                                                       sizeof(T) * 8,
                                                                       stream));
  } else {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr,
                                                             temp_storage_bytes,
                                                             d_keys_in,
                                                             (T*)nullptr,
                                                             d_values_in,
                                                             (int*)nullptr,
                                                             num_items,
                                                             num_segments,
                                                             d_offsets,
                                                             d_offsets + 1,
                                                             0,
                                                             sizeof(T) * 8,
                                                             stream));
  }
}

template void GetTempStorageSize(
    const float* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

template void GetTempStorageSize(
    const half* d_keys_in,
    const int* d_values_in,
    int* d_offsets,
    int num_items,
    int num_segments,
    hipStream_t stream,
    bool is_descending,
    size_t& temp_storage_bytes);

// TODO: merge to one kernel
__global__ void SetupParamsKernel(int* d_values_in,
                                  int* d_offsets,
                                  int batch_size,
                                  int vocab_size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = batch_size * vocab_size;
  if (index < total_elements) {
    d_values_in[index] = index % vocab_size;
  }
  if (index < batch_size + 1) {
    d_offsets[index] = index * vocab_size;
  }
}

void LaunchSetupParamsKernel(int* d_values_in,
                             int* d_offsets,
                             int batch_size,
                             int vocab_size,
                             hipStream_t stream) {
  int total_elements = batch_size * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  SetupParamsKernel<<<gridSize, blockSize, 0, stream>>>(d_values_in,
                                                        d_offsets,
                                                        batch_size,
                                                        vocab_size);
}

template <typename T>
void LaunchSortPairs(void* d_temp_storage,
                     size_t temp_storage_bytes,
                     const T* d_keys_in,
                     T* d_keys_out,
                     const int* d_values_in,
                     int* d_values_out,
                     int num_items,
                     int num_segments,
                     int* d_offsets,
                     hipStream_t stream,
                     bool is_descending) {
  if (is_descending) {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage,
                                                                       temp_storage_bytes,
                                                                       d_keys_in,
                                                                       d_keys_out,
                                                                       d_values_in,
                                                                       d_values_out,
                                                                       num_items,
                                                                       num_segments,
                                                                       d_offsets,
                                                                       d_offsets + 1,
                                                                       0,
                                                                       sizeof(T) * 8,
                                                                       stream));
  } else {
    CUDA_CALL_THROW(hipcub::DeviceSegmentedRadixSort::SortPairs(d_temp_storage,
                                                             temp_storage_bytes,
                                                             d_keys_in,
                                                             d_keys_out,
                                                             d_values_in,
                                                             d_values_out,
                                                             num_items,
                                                             num_segments,
                                                             d_offsets,
                                                             d_offsets + 1,
                                                             0,
                                                             sizeof(T) * 8,
                                                             stream));
  }
}

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const float* d_keys_in,
                              float* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

template void LaunchSortPairs(void* d_temp_storage,
                              size_t temp_storage_bytes,
                              const half* d_keys_in,
                              half* d_keys_out,
                              const int* d_values_in,
                              int* d_values_out,
                              int num_items,
                              int num_segments,
                              int* d_offsets,
                              hipStream_t stream,
                              bool is_descending);

// A stateful callback functor that maintains a running prefix to be applied
// during consecutive scan operations.
struct BlockPrefixCallbackOp {
  float running_total;  // running prefix

  __device__ BlockPrefixCallbackOp(float running_total) : running_total(running_total) {}
  // Callback operator to be entered by the first warp of threads in the block.
  // Thread-0 is responsible for returning a value for seeding the block-wide scan.
  __device__ float operator()(float block_aggregate) {
    float old_prefix = running_total;
    running_total += block_aggregate;
    return old_prefix;
  }
};

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernelCustom(float* d_sorted_logits_in,
                                         const int* d_sorted_indices,
                                         T* d_logits_in_out,
                                         float top_p_threshold,
                                         float filter_value,
                                         int batch_size,
                                         int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).ExclusiveSum(sum, sum, prefix_op);

    __syncthreads();
    if (sum >= top_p_threshold) {
      int original_index = offset + d_sorted_indices[offset + idx];
      d_logits_in_out[original_index] = (T)filter_value;
    }
  }
}

template <typename T, int kBlockSize>
__global__ void FilterLogitsKernel(float* d_sorted_logits_in,
                                   const int* d_sorted_indices,
                                   T* d_logits_in_out,
                                   float top_p_threshold,
                                   float filter_value,
                                   int min_tokens_to_keep,
                                   int batch_size,
                                   int vocab_size) {
  int vocab_idx = threadIdx.x;
  int batch_id = blockIdx.x;
  int offset = batch_id * vocab_size;

  typedef hipcub::BlockScan<float, kBlockSize> BlockScan;
  __shared__ typename BlockScan::TempStorage temp_storage;
  BlockPrefixCallbackOp prefix_op(0);

  for (int idx = vocab_idx; idx < vocab_size; idx += kBlockSize) {
    float sum = d_sorted_logits_in[offset + idx];
    BlockScan(temp_storage).InclusiveSum(sum, sum, prefix_op);

    __syncthreads();

    if (sum <= top_p_threshold) {
      if (idx + min_tokens_to_keep < vocab_size) {
        int original_index = offset + d_sorted_indices[offset + idx];
        d_logits_in_out[original_index] = (T)filter_value;
      }
    }
  }
}

template <typename T>
void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                              const int* d_sorted_indices,
                              T* d_logits_in_out,
                              float top_p,
                              float filter_value,
                              int min_tokens_to_keep,
                              int batch_size,
                              int vocab_size,
                              hipStream_t stream,
                              bool is_descending) {
  constexpr int kBlockSize = 256;

  if (is_descending) {
    FilterLogitsKernelCustom<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                                   d_sorted_indices,
                                                                                   d_logits_in_out,
                                                                                   top_p,
                                                                                   filter_value,
                                                                                   batch_size,
                                                                                   vocab_size);
  } else {
    FilterLogitsKernel<T, kBlockSize><<<batch_size, kBlockSize, 0, stream>>>(d_sorted_logits_in,
                                                                             d_sorted_indices,
                                                                             d_logits_in_out,
                                                                             1 - top_p,
                                                                             filter_value,
                                                                             min_tokens_to_keep,
                                                                             batch_size,
                                                                             vocab_size);
  }
}

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       float* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

template void LaunchFilterLogitsKernel(float* d_sorted_logits_in,
                                       const int* d_sorted_indices,
                                       half* d_logits_in_out,
                                       float top_p,
                                       float filter_value,
                                       int min_tokens_to_keep,
                                       int batch_size,
                                       int vocab_size,
                                       hipStream_t stream,
                                       bool is_descending);

// Ref: https://github.com/pytorch/pytorch/blob/release/1.13/aten/src/ATen/native/cuda/MultinomialKernel.cu
template <typename scalar_t, typename accscalar_t>
__global__ void sampleMultinomialOnce(int32_t* dest,
                                      int distributions,
                                      int categories,
                                      scalar_t* sampled,
                                      scalar_t* dist,
                                      int stride_dist,        // dist->stride(0)
                                      int stride_categories,  // dist->stride(1)
                                      int* d_presence_mask) {
  extern __shared__ unsigned char my_smem[];
  __shared__ bool found;
  __shared__ unsigned foundPos;
  accscalar_t* smem = reinterpret_cast<accscalar_t*>(my_smem);
  accscalar_t accZero = static_cast<accscalar_t>(0);
  scalar_t zero = static_cast<scalar_t>(0);
  for (int curDist = blockIdx.x;
       curDist < distributions; curDist += gridDim.x) {
    // Assume sum = 1 in Top P sampling as the input is softmaxed.
    accscalar_t sum = 1;

    // Broadcast sum and sample value
    if (threadIdx.x == 0) {
      // Make sure the sum of our distribution didn't overflow
      // CUDA_KERNEL_ASSERT(!_isinf(val));
      // CUDA_KERNEL_ASSERT(sum > accZero);
      foundPos = 0;
      smem[0] = sum;
      smem[1] = sampled[curDist];
    }
    __syncthreads();
    sum = smem[0];
    scalar_t sample = static_cast<scalar_t>(smem[1]);
    __syncthreads();
    if (sum == accZero) {
      // Choose the first element
      if (threadIdx.x == 0) {
        dest[curDist] = 0;
      }
      continue;
    }
    int chunks = (categories + (int)blockDim.x - 1) / blockDim.x;
    accscalar_t prevHighProb = accZero;
    found = false;
    for (int chunk = 0; chunk < chunks && !found; ++chunk) {
      // All threads in bounds load a value
      int cat = chunk * blockDim.x + threadIdx.x;
      accscalar_t dist_val = cat < categories ? static_cast<accscalar_t>(dist[curDist * stride_dist + cat * stride_categories]) / sum : accZero;
      smem[threadIdx.x] = dist_val;
      __syncthreads();
      // Perform an inclusive prefix sum of the shared memory contents
      for (int offset = 1; offset < blockDim.x; offset *= 2) {
        accscalar_t val = accZero;
        if (threadIdx.x >= offset) {
          val = smem[threadIdx.x - offset] + smem[threadIdx.x];
        }
        __syncthreads();
        if (threadIdx.x >= offset) {
          smem[threadIdx.x] = val;
        }
        __syncthreads();
      }
      // Each thread will check to see if the sample falls in its bucket
      scalar_t curBucket =
          static_cast<scalar_t>(smem[threadIdx.x] + prevHighProb);
      scalar_t prevBucket = static_cast<scalar_t>(
          threadIdx.x == 0 ? prevHighProb
                           : smem[threadIdx.x - 1] + prevHighProb);
      bool inBucket =
          (cat < categories) &&
          (!(sample >= curBucket) &&
           (sample >= prevBucket) &&
           (dist_val > zero));
      if (inBucket) {
        // We're done; we have the sample
        // Torch indices are 1-based
        atomicMax(&foundPos, cat);
        found = true;
      }
      // Store the previous scan's high value for future use
      prevHighProb = prevHighProb + smem[blockDim.x - 1];
      __syncthreads();
    }
    if (threadIdx.x == 0) {
      if (found) {
        dest[curDist] = foundPos;
      } else {
        // This should address a rare bug where we don't select a valid index. This likely occurs when
        // due to floating point arithmetic rounding errors, our cumulative sum does not add up to 1, but
        // and our uniform sample is greater than this value. In this case we likely have unitialized memory
        // in dest[curDist]. So basically we will loop through the distribution and pick the largest index
        // where the distribution is non-zero. This is obviously terribly inefficient, but due to the
        // rarity in which this occurs, this should not be an issue.
        for (int cat = categories - 1; cat >= 0; --cat) {
          if (dist[curDist * stride_dist + cat * stride_categories] > zero) {
            dest[curDist] = cat;
            break;
          }
        }
      }
    }
  }

  // update presence mask
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index >= distributions * categories) {
    return;
  }
  int dist_idx = index / categories;
  int cat_idx = index % categories;
  if (dest[dist_idx] == cat_idx) {
    d_presence_mask[index] = 1;
  }
}

// Only support n_sample = 1
void TorchMultinomialKernelLauncher(float* d_input,
                                    float* d_sampled,
                                    int32_t* d_output,
                                    int batch_size,
                                    int vocab_size,
                                    int* d_presence_mask,
                                    hipStream_t stream) {
  // Store the props in class variables
  int device;
  CUDA_CALL_THROW(hipGetDevice(&device));
  hipDeviceProp_t props;
  CUDA_CALL_THROW(hipGetDeviceProperties(&props, device));

  int numSM = props.multiProcessorCount;
  int maxThreads = props.maxThreadsPerBlock;
  int warp_size = 32;  // at::cuda::warp_size();
  int requiredWarps = (vocab_size + warp_size - 1) / warp_size;
  int requiredThreads = std::min(maxThreads, requiredWarps * warp_size);
  int requiredShared = requiredThreads * sizeof(float);

  dim3 block(requiredThreads);
  dim3 grid(std::min(batch_size, numSM * 4));

  sampleMultinomialOnce<float, float>
      <<<grid, block, requiredShared, stream>>>(d_output,
                                                batch_size,
                                                vocab_size,
                                                d_sampled,
                                                d_input,
                                                vocab_size,
                                                1,
                                                d_presence_mask);
}

__global__ void UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel(int32_t* tgt_indir_cache,
                                                                            const int32_t* src_indir_cache,
                                                                            const int32_t* beam_ids,
                                                                            int batch_size,
                                                                            int beam_width,
                                                                            int input_seq_length,
                                                                            int max_seq_length,
                                                                            int current_length) {
  int time_step = threadIdx.x + blockIdx.x * blockDim.x;
  int bb_id = threadIdx.y + blockIdx.y * blockDim.y;
  const int batch_id = bb_id / beam_width;
  const int beam_id = bb_id % beam_width;

  if (bb_id >= beam_width * batch_size || time_step >= current_length) {
    return;
  }

  const int src_beam = beam_ids[batch_id * beam_width + beam_id] % beam_width;

  const int tgt_offset = batch_id * beam_width * max_seq_length + beam_id * max_seq_length + time_step;

  if (time_step < input_seq_length) {
    // For time steps that correspond to the input sequence,
    // the beam that it comes from is always 0.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(0);
  } else if (time_step == (current_length - 1)) {
    // For the final (newly generated) time step,
    // the beam that it comes from is always the beam that we
    // are currently processing (i.e.) from this point on, these time-steps
    // form the new beams.
    tgt_indir_cache[tgt_offset] = static_cast<int32_t>(beam_id);
  } else {
    // For all other time-steps, we look up the source indirection, to
    // see which beam it came from based on the `src_beam`.
    const int src_offset = batch_id * beam_width * max_seq_length + src_beam * max_seq_length + time_step;
    tgt_indir_cache[tgt_offset] = src_indir_cache[src_offset];
  }
}

void UpdateDecoderMaskedMultiHeadAttentionCacheIndirection(int32_t* tgt_indir_cache,
                                                           const int32_t* src_indir_cache,
                                                           const int32_t* beam_ids,
                                                           int batch_size,
                                                           int beam_width,
                                                           int input_seq_length,
                                                           int max_seq_length,
                                                           int current_length,
                                                           hipStream_t stream) {
  const dim3 block(32);
  const dim3 grid((current_length + block.x - 1) / block.x, batch_size * beam_width);
  UpdateDecoderMaskedMultiHeadAttentionCacheIndirectionKernel<<<grid, block, 0, stream>>>(tgt_indir_cache,
                                                                                          src_indir_cache,
                                                                                          beam_ids,
                                                                                          batch_size,
                                                                                          beam_width,
                                                                                          input_seq_length,
                                                                                          max_seq_length,
                                                                                          current_length);
}

#ifndef USE_ROCM
namespace {
template <typename T, size_t size>
struct TypeMapper : public V_vec_m_<T, size> {};

template <>
struct TypeMapper<int32_t, 2> {
  using Type = uint2;
};

template <>
struct TypeMapper<int32_t, 4> {
  using Type = uint4;
};
}  // namespace
#endif

template <typename T>
__global__ void KeyCacheExpansionKernel(const T* input,
                                        T* output,
                                        int beam_width,
                                        int max_seq_length,
                                        int head_size) {
  const int num_heads = gridDim.y;
  const int sequence_length = gridDim.z;

  const int bbid = blockIdx.x;
  const int batch_id = bbid / beam_width;
  const int head_id = blockIdx.y;
  const int s = blockIdx.z;
  const int tidx = threadIdx.x;

  const int input_offset = ((batch_id * num_heads + head_id) * sequence_length + s) * head_size + tidx;
  const int output_offset = ((bbid * num_heads + head_id) * max_seq_length + s) * head_size + tidx;

  if (tidx < head_size) {
    output[output_offset] = input[input_offset];
  }
}

template <typename T>
void KeyCacheExpansionKernelLauncher(const T* key_cache,
                                     T* key_cache_expanded,
                                     int batch_size,
                                     int beam_width,
                                     int num_heads,
                                     int sequence_length,
                                     int max_seq_length,
                                     int head_size,
                                     hipStream_t stream) {
  const dim3 grid(batch_size * beam_width, num_heads, sequence_length);

  int equiv_head_size = (head_size & 1) == 0 ? (head_size >> 1) : head_size;
  equiv_head_size = (equiv_head_size & 1) == 0 ? (equiv_head_size >> 1) : equiv_head_size;

  // Here we know head_size is smaller than max_thread_num_per_block
  int tpb = std::max(32, equiv_head_size);

  // round up tpb to power of 2
  --tpb;
  tpb |= (tpb >> 1);
  tpb |= (tpb >> 2);
  tpb |= (tpb >> 4);
  tpb |= (tpb >> 8);
  tpb |= (tpb >> 16);
  tpb++;

#ifndef USE_ROCM
  if ((head_size % 4) == 0) {
    using vec_type = typename TypeMapper<T, 4>::Type;
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(key_cache),
                                                        reinterpret_cast<vec_type*>(key_cache_expanded),
                                                        beam_width,
                                                        max_seq_length,
                                                        equiv_head_size);
  } else if ((head_size & 1) == 0) {
    using vec_type = typename TypeMapper<T, 2>::Type;
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(key_cache),
                                                        reinterpret_cast<vec_type*>(key_cache_expanded),
                                                        beam_width,
                                                        max_seq_length,
                                                        equiv_head_size);
  } else {
#endif
    const dim3 block(tpb);
    KeyCacheExpansionKernel<<<grid, block, 0, stream>>>(key_cache,
                                                        key_cache_expanded,
                                                        beam_width,
                                                        max_seq_length,
                                                        head_size);
#ifndef USE_ROCM
  }
#endif
}

template void KeyCacheExpansionKernelLauncher(const float* key_cache,
                                              float* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template void KeyCacheExpansionKernelLauncher(const half* key_cache,
                                              half* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template void KeyCacheExpansionKernelLauncher(const int32_t* key_cache,
                                              int32_t* key_cache_expanded,
                                              int batch_size,
                                              int beam_width,
                                              int num_heads,
                                              int sequence_length,
                                              int max_seq_length,
                                              int head_size,
                                              hipStream_t stream);

template <typename T>
__global__ void BufferExpansionKernel(const T* input,
                                      T* output,
                                      int chunk_size) {
  const int batch_id = blockIdx.x;
  const int beam_id = blockIdx.y;
  const int tidx = threadIdx.x;
  const int beam_size = gridDim.y;
  const int idx = blockIdx.z * blockDim.x + tidx;

  const int input_offset = batch_id * chunk_size + idx;
  const int output_offset = batch_id * beam_size * chunk_size + beam_id * chunk_size + idx;

  if (idx < chunk_size) {
    output[output_offset] = input[input_offset];
  }
}

template <typename T>
void BufferExpansionKernelLauncher(const T* input,
                                   T* output,
                                   int batch_size,
                                   int beam_width,
                                   int chunk_size,
                                   hipStream_t stream) {
  const dim3 block(128);

#ifndef USE_ROCM
  if ((chunk_size % 4) == 0) {
    using vec_type = typename TypeMapper<T, 4>::Type;
    const dim3 grid(batch_size, beam_width, (chunk_size / 4 + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(input),
                                                      reinterpret_cast<vec_type*>(output),
                                                      chunk_size / 4);
  } else if ((chunk_size & 1) == 0) {
    using vec_type = typename TypeMapper<T, 2>::Type;
    const dim3 grid(batch_size, beam_width, (chunk_size / 2 + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(reinterpret_cast<const vec_type*>(input),
                                                      reinterpret_cast<vec_type*>(output),
                                                      chunk_size / 2);
  } else {
#endif
    const dim3 grid(batch_size, beam_width, (chunk_size + block.x - 1) / block.x);
    BufferExpansionKernel<<<grid, block, 0, stream>>>(input,
                                                      output,
                                                      chunk_size);
#ifndef USE_ROCM
  }
#endif
}

template void BufferExpansionKernelLauncher(const float* input,
                                            float* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

template void BufferExpansionKernelLauncher(const half* input,
                                            half* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

template void BufferExpansionKernelLauncher(const int32_t* input,
                                            int32_t* output,
                                            int batch_size,
                                            int beam_width,
                                            int chunk_size,
                                            hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
