#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "beam_search_impl.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace contrib {
namespace cuda {

__global__ void NextTokenKernel(const int64_t* next_token_indices,
                                int64_t* next_indices,
                                int64_t* next_tokens,
                                int vocab_size,
                                int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    next_indices[index] = next_token_indices[index] / vocab_size;
    next_tokens[index] = next_token_indices[index] % vocab_size;
  }
}

/* NextToken kernel is corresponding to logic like the following:
   for i in range (batch_size):
    for j in range (top_k):
      next_indices[i, j] = next_token_indices[i, j] / vocab_size
      next_tokens[i, j] = next_token_indices[i, j] % vocab_size
*/
void LaunchNextTokenKernel(const int64_t* next_token_indices,
                           int64_t* next_indices,
                           int64_t* next_tokens,
                           int batch_size,
                           int top_k,
                           int vocab_size,
                           hipStream_t stream) {
  int total_elements = batch_size * top_k;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  NextTokenKernel<<<gridSize, blockSize, 0, stream>>>(next_token_indices, next_indices, next_tokens, vocab_size, total_elements);
}

__global__ void InitKernel(float* beam_scores,
                           int num_beams,
                           int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_elements) {
    int beam_index = index % num_beams;
    beam_scores[index] = beam_index > 0 ? static_cast<float>(-1e9) : 0.0f;  // This value exceeds limit of MLFloat16 so it is for float only.
  }
}

void LaunchInitKernel(
    float* beam_scores,
    int batch_size,
    int num_beams,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  InitKernel<<<gridSize, blockSize, 0, stream>>>(beam_scores, num_beams, total_elements);
}

template <typename T>
__global__ void VocabMaskKernel(T* log_probs,
                                const int* vocab_mask,
                                int vocab_size,
                                int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int word_id = index % vocab_size;

  if (index < total_elements && vocab_mask[word_id] == 0) {
    log_probs[index] = std::numeric_limits<T>::lowest();
  }
}

template <typename T>
void LaunchVocabMaskKernel(
    T* log_probs,
    const int* vocab_mask,
    int batch_size,
    int num_beams,
    int vocab_size,
    hipStream_t stream) {
  int total_elements = batch_size * num_beams * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  VocabMaskKernel<float><<<gridSize, blockSize, 0, stream>>>(log_probs, vocab_mask, vocab_size, total_elements);
}

// Instantiation
template void LaunchVocabMaskKernel(
    float* log_probs,
    const int* vocab_mask,
    int batch_size,
    int num_beams,
    int vocab_size,
    hipStream_t stream);

template <typename T>
__global__ void AddProbsKernel(T* log_probs,
                               T* cum_log_probs,
                               const int vocab_size,
                               const int total_elements) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int batch_beam_index = index / vocab_size;

  if (index < total_elements)
    log_probs[index] += cum_log_probs[batch_beam_index];
}

void LaunchAddProbsKernel(float* log_probs,
                          float* cum_log_probs,
                          const int batch_size,
                          const int num_beams,
                          const int vocab_size,
                          hipStream_t stream) {
  int total_elements = batch_size * num_beams * vocab_size;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  AddProbsKernel<float><<<gridSize, blockSize, 0, stream>>>(log_probs, cum_log_probs, vocab_size, total_elements);
}

template <typename T>
__global__ void UpdateInputsKernel(const T* old_mask_data,
                                   T* mask_data,
                                   int64_t* next_positions,
                                   int batch_beam_size,
                                   int current_length) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < batch_beam_size * current_length) {
    // Update attention mask like the following:
    //   for (int i = 0; i < batch_beam_size; i++) {
    //     for (int j = 0; j < current_length - 1; j++) {
    //       mask_data[i * current_length + j] = old_mask_data[i * (current_length - 1) + j];
    //     }
    //     mask_data[i * current_length + current_length - 1] = 1.0f;
    //   }
    int i = index / current_length;
    int j = index % current_length;
    mask_data[index] = (j < current_length - 1) ? old_mask_data[i * (current_length - 1) + j] : static_cast<T>(1.0f);

    // Update sequence length (or next positions) like the following:
    //   for (int i = 0; i < batch_beam_size; i++) {
    //     next_positions[i]++;
    //   }
    if (index < batch_beam_size) {
      next_positions[index]++;
    }
  }
}

template <typename T>
void LaunchUpdateKernel(const T* old_mask_data,
                        T* mask_data,
                        int64_t* next_positions,
                        int batch_beam_size,
                        int current_length,
                        hipStream_t stream) {
  assert(current_length > 0);
  int total_elements = batch_beam_size * current_length;
  constexpr int blockSize = 256;
  const int gridSize = (total_elements + blockSize - 1) / blockSize;
  UpdateInputsKernel<T><<<gridSize, blockSize, 0, stream>>>(old_mask_data, mask_data, next_positions, batch_beam_size, current_length);
}

// Instantiation
template void LaunchUpdateKernel(const float* old_mask_data,
                                 float* mask_data,
                                 int64_t* next_positions,
                                 int batch_beam_size,
                                 int current_length,
                                 hipStream_t stream);

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime