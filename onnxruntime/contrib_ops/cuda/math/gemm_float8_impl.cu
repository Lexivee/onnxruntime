#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "gemm_float8_impl.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_utils.h"
#include <algorithm>
#include <hip/hip_runtime.h>

namespace onnxruntime {
namespace contrib {
namespace cuda {

static const char* cublasGetErrorEnum(hipblasStatus_t error) {
  switch (error) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";

    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";

    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";

    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";

    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";

    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";

    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";

    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";

    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";

    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";

    default:
      return "<unknown>";
  }
}

void GemmFloat8_Impl::set(int M, int N, int K, int& lda, int& ldb, int& ldd) const {
  if (trans_A_ && !trans_B_) {  // TN
    lda = K;
    ldb = K;
    ldd = M;
  } else if (!trans_A_ && !trans_B_) {  // NN
    lda = M;
    ldb = K;
    ldd = M;
  } else if (!trans_A_ && trans_B_) {  // NT
    lda = M;
    ldb = N;
    ldd = M;
  } else {  // TT
    ORT_THROW("trans_A_ == true && trans_B_ == true not allowed.");
  }
}

onnxruntime::Status GemmFloat8_Impl::CudaCompute(
    const int32_t* dtypes, hipStream_t stream, hipblasLtHandle_t handle,
    const Tensor* A, const Tensor* B, const Tensor* C, Tensor* D,
    int M, int N, int K) const {
  int lda, ldb, ldd;
  set(M, N, K, lda, ldb, ldd);

  bool has_C = beta_ != 0 && C != nullptr;

  // broadcast bias if needed and is present
  if (has_C) {
    auto& a_shape = A->Shape();
    auto& b_shape = B->Shape();
    auto& c_shape = C->Shape();
    if (c_shape.Size() == 1) {
      // if C is (), (1,) or (1, 1), broadcast the scalar
      ORT_THROW("Broadcasting is not implemented in GemmFloat8.");
    } else if (c_shape.NumDimensions() == 1 || c_shape[0] == 1) {
      // C is (N,) or (1, N), broadcast using Y(N,M) = 1 * C(N,1) x ones(1,M) + 0 * C
      ORT_THROW("Broadcasting is not implemented in GemmFloat8.");
    } else if (b_shape.NumDimensions() == 2 && b_shape[1] == 1) {
      // B is (M, 1), broadcast using Y(N,M) = 1 * ones(N,1) x B(1,M) + 0 * C
      ORT_THROW("Broadcasting is not implemented in GemmFloat8.");
    } else {
      // C is (M, N), no broadcast needed.
      /*
      constexpr bool same_type = std::same_type<DType, BiasType>::value;
      if (same_type) {
        CUDA_RETURN_IF_ERROR(hipMemcpyAsync(out_data, b_data, static_cast<size_t>(M) * N * sizeof(T), hipMemcpyDeviceToDevice, Stream(ctx)));
      }
      */
    }
  }

  // Gemm, note that CUDA assumes col-major, so Y(N,M) = alpha * op(B) x op(A) + beta * C
  std::cout << "GemmF8-1\n";

  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr, Ddesc = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};

  hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  std::cout << "GemmF8-2\n";

  // Create matrix descriptors. Not setting any extra attributes.
  hipDataType atype = ToCudaDataType(dtypes[0]);
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Adesc, atype, trans_A_ ? M : K, trans_A_ ? K : M, lda));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Bdesc, ToCudaDataType(dtypes[1]), trans_B_ ? K : N, trans_B_ ? N : K, ldb));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Ddesc, ToCudaDataType(dtypes[3]), M, N, ldd));

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));

  // HIP_R_32F is the scale type for the time being since it is not used.
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulDescCreate#cublasltmatmuldesccreate
  hipblasLtMatmulDescCreate(&operationDesc, compute_type_, scale_type_);
  hipblasOperation_t transa = trans_A_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb = trans_B_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb));
  const int8_t ifast_accumulation_mode = fast_accumulation_mode_ ? 0 : 1;
  hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &ifast_accumulation_mode, sizeof(ifast_accumulation_mode));

  std::cout << "GemmF8-3\n";
  /*
  // TODO add inputs for the scales.
  // No scale for the time being so no need to set.
  HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER,
  HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER,
  HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER
  HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER
  HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER
  */

  if (sm_count_ != 0) {
    int math_sm_count = sm_count_;
    hipblasLtMatmulDescSetAttribute(
        operationDesc, CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET,
        &math_sm_count, sizeof(math_sm_count));
  }

  std::cout << "GemmF8-4\n";
  if (has_C) {
    std::cout << "GemmF8-5\n";
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Cdesc, ToCudaDataType(dtypes[2]), M, N, ldd));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  }
  /*
  // No bias for the time being.
  if (relu_bias) {
    std::cout << "GemmF8-6\n";
    hipDataType bias_type = ToCudaDataType(dtypes[4]);
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                          HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE,
                                                          &bias_type, sizeof(bias_type)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                          HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                                          relu_bias, sizeof(*relu_bias)));
    epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }
  */

  std::cout << "GemmF8-7\n";
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));

  hipblasLtMatmulPreferenceCreate(&preference);

  // See https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true with H100).
  size_t type_size = std::max(std::max(TypeSize(dtypes[0]), TypeSize(dtypes[1])), std::max(std::max(TypeSize(dtypes[2]), TypeSize(dtypes[3])), TypeSize(dtypes[4])));
  size_t workspaceSize = std::max(K * M, K * N) * type_size;  // suggested fixed value 24Mb
  hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

  std::cout << "GemmF8-8\n";
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic
  int returnedResults = 0;
  hipblasStatus_t cuda_status = hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Cdesc,
                                                              Ddesc, preference, 1, &heuristicResult, &returnedResults);
  ORT_ENFORCE(returnedResults > 0 && cuda_status == HIPBLAS_STATUS_SUCCESS,
              "Unable to find any suitable algorithm due to ", cublasGetErrorEnum(cuda_status),
              ", preference=", preference, ", returnedResults=", returnedResults,
              ", A_type=", ToCudaDataType(dtypes[0]), ", B_type=", ToCudaDataType(dtypes[1]),
              ", C_type=", ToCudaDataType(dtypes[2]), ", D_type=", ToCudaDataType(dtypes[3]),
              ", bias_type=", ToCudaDataType(dtypes[4]), ", computeType=", compute_type_,
              ", transA=", trans_A_, ", transB=", trans_B_,
              ", M=", M, ", N=", N, ", K=", K, ", lda=", lda, ", ldb=", ldb, ", ldd=", ldd,
              ", workspaceSize=", workspaceSize, ". Check NVDIDIA documentation to see what combination is valid: ",
              "https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic.");
  std::cout << "GemmF8-9\n";
  void* workspace = nullptr;
  CUDA_CALL_THROW(hipMalloc((void**)&workspace, workspaceSize));
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmul#cublasltmatmul
  hipblasLtMatmul(handle,
                 operationDesc,
                 static_cast<const void*>(&alpha_), /* alpha */
                 A,                                 /* A */
                 Adesc,
                 B, /* B */
                 Bdesc,
                 static_cast<const void*>(&beta_), /* beta */
                 C,                                /* C */
                 Cdesc,
                 D, /* D */
                 Ddesc,
                 &heuristicResult.algo, /* algo */
                 workspace,             /* workspace */
                 workspaceSize,
                 stream); /* stream */
  std::cout << "GemmF8-10\n";
  hipFree(workspace);

  std::cout << "GemmF8-11\n";
  hipblasLtMatmulPreferenceDestroy(preference);
  if (Cdesc != nullptr && Cdesc != Ddesc)
    hipblasLtMatrixLayoutDestroy(Cdesc);
  hipblasLtMatrixLayoutDestroy(Ddesc);
  hipblasLtMatrixLayoutDestroy(Bdesc);
  hipblasLtMatrixLayoutDestroy(Adesc);
  hipblasLtMatmulDescDestroy(operationDesc);
  std::cout << "GemmF8-12\n";
  return onnxruntime::Status::OK();
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
