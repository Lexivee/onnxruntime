#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "gemm_float8.cuh"

namespace onnxruntime {
namespace cuda {

template <typename AType, typename BType, typename CType, typename DType, typename BiasType>
void GemmFloat8_Impl<AType, BType, CType, DType, BiasType>::CudaCompute() const {
  // broadcast bias if needed and is present
  if (beta_ != 0 && C != nullptr) {
    auto& c_shape = C->Shape();
    const CudaT* b_data = reinterpret_cast<const CudaT*>(C->Data<CType>());
    if (c_shape.Size() == 1) {
      // if C is (), (1,) or (1, 1), broadcast the scalar
      ORT_RAISE("Broadcasting is not implemented in GemmFloat8.");
    } else if (c_shape.NumDimensions() == 1 || c_shape[0] == 1) {
      // C is (N,) or (1, N), broadcast using Y(N,M) = 1 * C(N,1) x ones(1,M) + 0 * C
      ORT_RAISE("Broadcasting is not implemented in GemmFloat8.");
    } else if (b_shape.NumDimensions() == 2 && b_shape[1] == 1) {
      // B is (M, 1), broadcast using Y(N,M) = 1 * ones(N,1) x B(1,M) + 0 * C
      ORT_RAISE("Broadcasting is not implemented in GemmFloat8.");
    } else {
      // C is (M, N), no broadcast needed.
      /*
      constexpr bool same_type = std::same_type<DType, BiasType>::value;
      if (same_type) {
        CUDA_RETURN_IF_ERROR(hipMemcpyAsync(out_data, b_data, static_cast<size_t>(M) * N * sizeof(T), hipMemcpyDeviceToDevice, Stream(ctx)));
      }
      */
    }
  }

  CudaDType alpha = alpha_cast_;
  CudaDType beta = beta_cast_;
  // Gemm, note that CUDA assumes col-major, so Y(N,M) = alpha * op(B) x op(A) + beta * C

  constexpr auto A_type = ToCudaDataType<AType>();
  constexpr auto B_type = ToCudaDataType<BType>();
  constexpr auto C_type = ToCudaDataType<CType>();
  constexpr auto D_type = ToCudaDataType<DType>();
  constexpr auto bias_type = ToCudaDataType<BiasType>();

  // It should be true all the time unless we extend the definition of the kernel to other combinations.
  /*
  constexpr hipblasComputeType_t gemm_compute_type = (A_type == HIP_R_8F_E5M2 || B_type == HIP_R_8F_E5M2)
                                          ? HIPBLAS_COMPUTE_32F
                                          : HIPBLAS_COMPUTE_32F_FAST_TF32;
  */

  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr, Ddesc = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create matrix descriptors. Not setting any extra attributes.
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Adesc, A_type, trans_A_ ? M : K, trans_A_ ? K : M, lda));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Bdesc, B_type, trans_B_ ? K : N, trans_B_ ? N : K, ldb));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Ddesc, D_type, m, n, ldd));

  // HIP_R_32F is the scale type for the time being since it is not used.
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescCreate(&operationDesc, compute_type_, HIP_R_32F));
  hipblasOperation_t transa = trans_A_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb = trans_B_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));
  const int8_t fast_accumulation_mode = fast_accumulation_mode_ ? 0 : 1;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc, CUBLASLT_MATMUL_DESC_FAST_ACCUM, &fastAccuMode, sizeof(fast_accumulation_mode)));

  /*
  // TODO add inputs for the scales.
  // No scale for the time being so no need to set.
  HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER,
  HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER,
  HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER
  HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER
  HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER
  */

  if (sm_count_ != 0) {
    int math_sm_count = sm_count_;
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET,
        &math_sm_count, sizeof(math_sm_count)));
  }

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(&Cdesc, C_type, M, N, ldd));
  if (bias) {
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                          HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE,
                                                          &bias_type, sizeof(bias_type)));
    epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                        HIPBLASLT_MATMUL_DESC_BIAS_POINTER,
                                                        &bias_ptr, sizeof(bias_ptr)));

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(operationDesc,
                                                        HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                                        &epilogue, sizeof(epilogue)));

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceCreate(&preference));

  // See https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true with H100).
  constexpr size_t type_size = std::max(sizeof(AType), sizof(BType), sizeof(CType), sizeof(DType), sizeof(BiasType));
  size_t workspaceSize = std::max(K * M, K * N) * type_size;  // suggested fixed value 24Mb
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceSetAttribute(
      preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
      &workspaceSize, sizeof(workspaceSize)));

  int returnedResults = 0;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulAlgoGetHeuristic(handle, operationDesc, Adesc, Bdesc, Cdesc,
                                                        Ddesc, preference, 1, &heuristicResult,
                                                        &returnedResults));
  ORT_ENFORCE(returnedResults > 0, "Unable to find any suitable algorithm.");
  void* workspace = hipMalloc(workspaceSize);
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmul(handle,
                                        operationDesc,
                                        static_cast<const void*>(&alpha_cast_), /* alpha */
                                        A,                                      /* A */
                                        Adesc,
                                        B, /* B */
                                        Bdesc,
                                        static_cast<const void*>(&beta_cast_), /* beta */
                                        C,                                     /* C */
                                        Cdesc,
                                        D, /* D */
                                        Ddesc,
                                        &heuristicResult.algo, /* algo */
                                        workspace,             /* workspace */
                                        workspaceSize,
                                        stream)); /* stream */
  hipFree(workspace);

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceDestroy(preference));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Ddesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Cdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Bdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Adesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescDestroy(operationDesc));
}

}  // namespace cuda
}  // namespace onnxruntime
