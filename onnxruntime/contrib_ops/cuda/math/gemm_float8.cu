// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "gemm_float8.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/providers/cuda/shared_inc/cuda_utils.h"
#include <algorithm>
#include <hip/hip_runtime.h>

namespace onnxruntime {
namespace contrib {
namespace cuda {

static const char* cublasGetErrorEnum(hipblasStatus_t error) {
  switch (error) {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
    default:
      return "<unknown>";
  }
}

static const char* CudaDataTypeToString(hipDataType dt) {
  switch (dt) {
    case HIP_R_16F:
      return "HIP_R_16F";
    case HIP_R_16BF:
      return "HIP_R_16BF";
    case HIP_R_32F:
      return "HIP_R_32F";
#if (CUDA_VERSION >= 11080)
    case HIP_R_8F_E4M3:
      return "HIP_R_8F_E4M3";
    case HIP_R_8F_E5M2:
      return "HIP_R_8F_E5M2";
#endif
    default:
      return "<unknown>";
  }
}

static const char* CublasComputeTypeToString(hipblasComputeType_t ct) {
  switch (ct) {
    case HIPBLAS_COMPUTE_16F:
      return "HIPBLAS_COMPUTE_16F";
    case HIPBLAS_COMPUTE_32F:
      return "HIPBLAS_COMPUTE_32F";
    case HIPBLAS_COMPUTE_32F_FAST_16F:
      return "HIPBLAS_COMPUTE_32F_FAST_16F";
    case HIPBLAS_COMPUTE_32F_FAST_16BF:
      return "HIPBLAS_COMPUTE_32F_FAST_16BF";
    case HIPBLAS_COMPUTE_32F_FAST_TF32:
      return "HIPBLAS_COMPUTE_32F_FAST_TF32";
    case HIPBLAS_COMPUTE_64F:
      return "HIPBLAS_COMPUTE_64F";
    default:
      return "<unknown>";
  }
}

// It must exist somewhere already.
hipDataType ToCudaDataType(int32_t element_type) {
  switch (element_type) {
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT:
      return HIP_R_32F;
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT16:
      return HIP_R_16F;
    case ONNX_NAMESPACE::TensorProto_DataType_BFLOAT16:
      return HIP_R_16BF;
#if (!defined(DISABLE_FLOAT8_TYPES) && (CUDA_VERSION >= 11080))
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT8E4M3FN:
      return HIP_R_8F_E4M3;
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT8E5M2:
      return HIP_R_8F_E5M2;
#endif
    default:
      ORT_THROW("Unexpected element_type=", element_type, ".");
  }
}

// It must exist somewhere already.
int32_t TypeSize(int32_t element_type) {
  switch (element_type) {
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT:
      return 4;
    case ONNX_NAMESPACE::TensorProto_DataType_BFLOAT16:
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT16:
      return 2;
#if (!defined(DISABLE_FLOAT8_TYPES) && (CUDA_VERSION >= 11080))
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT8E4M3FN:
    case ONNX_NAMESPACE::TensorProto_DataType_FLOAT8E5M2:
      return 1;
#endif
    default:
      ORT_THROW("Unexpected element_type=", element_type, ".");
  }
}

void GemmFloat8::set(const TensorShape& a_shape, const TensorShape& b_shape, int& M, int& N, int& K, int& lda, int& ldb, int& ldd, bool row_major) const {
  constexpr int ir = 0;
  constexpr int ic = 1 - ir;
  if (transA_ && !transB_) {  // TN
    M = a_shape[ic];
    N = b_shape[ic];
    K = a_shape[ir];
    lda = a_shape[row_major ? ic : ir];
    ldb = b_shape[row_major ? ic : ir];
    ldd = b_shape[row_major ? ic : ir];
  } else if (!transA_ && !transB_) {  // NN
    M = a_shape[ir];
    N = b_shape[ic];
    K = a_shape[ic];
    lda = a_shape[row_major ? ic : ir];
    ldb = b_shape[row_major ? ic : ir];
    ldd = b_shape[row_major ? ic : ir];
  } else if (!transA_ && transB_) {  // NT
    M = a_shape[ir];
    N = b_shape[ir];
    K = a_shape[ic];
    lda = a_shape[row_major ? ic : ir];
    ldb = b_shape[row_major ? ic : ir];
    ldd = b_shape[row_major ? ir : ic];
  } else {  // TT
    M = a_shape[ic];
    N = b_shape[ir];
    K = a_shape[ir];
    lda = a_shape[row_major ? ir : ic];
    ldb = b_shape[row_major ? ir : ic];
    ldd = b_shape[row_major ? ic : ir];
  }
}

template <typename TValue>
int32_t GetTypeAndShape(const TValue* input,
                        TensorShape& shape,
                        bool swap = false) {
  shape = input->Shape();
  ORT_ENFORCE(shape.NumDimensions() == 2);
  if (swap) {
    std::swap(shape[0], shape[1]);
  }
  return input->GetElementType();
}

Status GemmFloat8::ComputeInternal(OpKernelContext* ctx) const {
  const Tensor* input_A = nullptr;
  const Tensor* input_B = nullptr;
  const Tensor* input_C = nullptr;
  const Tensor* scale_A = nullptr;
  const Tensor* scale_B = nullptr;
  const Tensor* scale_Y = nullptr;
  bool has_scales = false;
  bool has_bias = false;
  int n_inputs = ctx->InputCount();
  ORT_ENFORCE(n_inputs >= 2, "A and/or B is missing.");
  input_A = ctx->Input<Tensor>(0);
  input_B = ctx->Input<Tensor>(1);
  if (n_inputs == 3) {
    input_C = ctx->Input<Tensor>(2);
    has_bias = true;
  } else if (n_inputs > 3) {
    ORT_ENFORCE(n_inputs == 6, "Unexpected number of inputs=", n_inputs, ".");
    has_scales = true;
    scale_A = ctx->Input<Tensor>(3);
    scale_B = ctx->Input<Tensor>(4);
    scale_Y = ctx->Input<Tensor>(5);
    ORT_ENFORCE(scale_A->GetElementType() == ONNX_NAMESPACE::TensorProto_DataType_FLOAT);
    ORT_ENFORCE(scale_B->GetElementType() == ONNX_NAMESPACE::TensorProto_DataType_FLOAT);
    ORT_ENFORCE(scale_Y->GetElementType() == ONNX_NAMESPACE::TensorProto_DataType_FLOAT);
    if (ctx->Input<Tensor>(2) != nullptr) {
      input_C = ctx->Input<Tensor>(2);
      has_bias = true;
      ORT_ENFORCE(input_C->GetElementType() == dtype_, "Bias type must be equal to dtype.");
    }
  }

  if (row_major_)
    return ComputeColMajor(ctx, n_inputs, has_bias, has_scales, input_A, input_B,
                           input_C, scale_A, scale_B, scale_Y);
  return ComputeRowMajor(ctx, n_inputs, has_bias, has_scales, input_A, input_B,
                         input_C, scale_A, scale_B, scale_Y);
}

Status GemmFloat8::ComputeRowMajor(
    OpKernelContext* ctx, int n_inputs, bool has_bias, bool has_scales,
    const Tensor* input_A, const Tensor* input_B,
    const Tensor* input_C, const Tensor* scale_A,
    const Tensor* scale_B, const Tensor* scale_Y) const {
  TensorShape shape_A, shape_B, shape_C, shape_Y;
  int32_t dtype_A, dtype_B, dtype_C, dtype_Y;
  dtype_A = GetTypeAndShape(input_A, shape_A);
  dtype_B = GetTypeAndShape(input_B, shape_B);

  int M, N, K, lda, ldb, ldd;
  set(shape_A, shape_B, M, N, K, lda, ldb, ldd, true);

  TensorShape dimensions{M, N};
  Tensor* Y = ctx->Output(0, dimensions);
  dtype_Y = GetTypeAndShape(Y, shape_Y);
  dtype_C = has_bias ? GetTypeAndShape(input_C, shape_C)
                     : ONNX_NAMESPACE::TensorProto_DataType_FLOAT;
  return ComputeGemm(ctx, n_inputs, has_bias, has_scales, dtype_A, dtype_B, dtype_C,
                     dtype_Y, shape_A, shape_B, shape_C, shape_Y, transA_, transB_,
                     input_A->DataRaw(), input_B->DataRaw(),
                     has_bias ? input_C->DataRaw() : nullptr,
                     has_scales ? scale_A->DataRaw() : nullptr,
                     has_scales ? scale_B->DataRaw() : nullptr,
                     has_scales ? scale_Y->DataRaw() : nullptr,
                     Y->MutableDataRaw(), M, N, K, lda, ldb, ldd);
}

Status GemmFloat8::ComputeColMajor(
    OpKernelContext* ctx, int n_inputs, bool has_bias, bool has_scales,
    const Tensor* input_A, const Tensor* input_B,
    const Tensor* input_C, const Tensor* scale_A,
    const Tensor* scale_B, const Tensor* scale_Y) const {
  TensorShape shape_A, shape_B, shape_C, shape_Y;
  int32_t dtype_A, dtype_B, dtype_C, dtype_Y;
  dtype_A = GetTypeAndShape(input_A, shape_A);
  dtype_B = GetTypeAndShape(input_B, shape_B);

  int M, N, K, lda, ldb, ldd;
  set(shape_A, shape_B, M, N, K, lda, ldb, ldd, true);

  std::swap(shape_A[0], shape_A[1]);
  std::swap(shape_B[0], shape_B[1]);

  TensorShape dimensions{M, N};
  Tensor* Y = ctx->Output(0, dimensions);
  dtype_Y = GetTypeAndShape(Y, shape_Y);
  dtype_C = has_bias ? GetTypeAndShape(input_C, shape_C, true)
                     : ONNX_NAMESPACE::TensorProto_DataType_FLOAT;

  return ComputeGemm(ctx, n_inputs, has_bias, has_scales, dtype_B, dtype_A, dtype_C,
                     dtype_Y, shape_B, shape_A, shape_C, shape_Y, transB_, transA_,
                     input_B->DataRaw(), input_A->DataRaw(),
                     has_bias ? input_C->DataRaw() : nullptr,
                     has_scales ? scale_B->DataRaw() : nullptr,
                     has_scales ? scale_A->DataRaw() : nullptr,
                     has_scales ? scale_Y->DataRaw() : nullptr,
                     Y->MutableDataRaw(), N, M, K, ldb, lda, ldd);
}

Status GemmFloat8::ComputeGemm(
    OpKernelContext* ctx, int n_inputs, bool has_bias, bool has_scales,
    int32_t dtype_A, int32_t dtype_B,
    int32_t dtype_C, int32_t dtype_Y,
    const TensorShape& shape_A, const TensorShape& shape_B,
    const TensorShape& shape_C, const TensorShape& shape_Y,
    bool trans_A, bool trans_B, const void* p_input_a, const void* p_input_b,
    const void* p_input_c, const void* p_scale_a, const void* p_scale_b,
    const void* p_scale_y, void* p_output_y, int M, int N, int K, int lda,
    int ldb, int ldd) const {
  hipStream_t stream = Stream(ctx);
  CUDA_RETURN_IF_ERROR(hipStreamSynchronize(stream));

  hipblasLtHandle_t cublasLt;
  CUBLAS_RETURN_IF_ERROR(hipblasLtCreate(&cublasLt));

  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr,
                         Ddesc = nullptr;

  // Create matrix descriptors. Not setting any extra attributes.
  hipDataType a_cuda_type = ToCudaDataType(dtype_A);
  hipDataType b_cuda_type = ToCudaDataType(dtype_B);
  hipDataType d_cuda_type = ToCudaDataType(dtype_Y);
  hipDataType scale_cuda_type =
      ToCudaDataType(ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT);
  hipDataType bias_cuda_type = ToCudaDataType(dtype_C);

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Adesc, a_cuda_type, trans_A ? K : M, trans_A ? M : K, lda));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Bdesc, b_cuda_type, trans_B ? N : K, trans_B ? K : N, ldb));
  CUBLAS_RETURN_IF_ERROR(
      hipblasLtMatrixLayoutCreate(&Ddesc, d_cuda_type, M, N, ldd));

  if (row_major_) {
    hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
  }

  CUBLAS_RETURN_IF_ERROR(
      hipblasLtMatmulDescCreate(&operationDesc, compute_type_, scale_cuda_type));
  hipblasOperation_t ctransa = trans_A ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t ctransb = trans_B ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &ctransa, sizeof(ctransa)));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &ctransb, sizeof(ctransb)));

  if (sm_count_ != 0) {
    int math_sm_count = static_cast<int>(sm_count_);
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET, &math_sm_count,
        sizeof(math_sm_count)));
  }

  if (has_scales) {
    // gemm float 8
    const int8_t ifast_accumulation_mode = fast_accumulation_mode_ ? 1 : 0;
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc,
        hipblasLtMatmulDescAttributes_t::CUBLASLT_MATMUL_DESC_FAST_ACCUM,
        &ifast_accumulation_mode, sizeof(ifast_accumulation_mode)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &p_scale_a,
        sizeof(p_scale_a)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &p_scale_b,
        sizeof(p_scale_b)));
    CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &p_scale_y,
        sizeof(p_scale_b)));

    // float 8
#if CUDA_VERSION >= 11080
    if (dtype_Y == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E4M3FN ||
        dtype_Y == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E5M2) {
      // For FP8 output, cuBLAS requires C_type to be same as bias_type
      CUBLAS_RETURN_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, bias_cuda_type, M, N, ldd));
      CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescSetAttribute(
          operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE, &bias_cuda_type,
          sizeof(bias_cuda_type)));
    } else {
      CUBLAS_RETURN_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
    }
  } else {
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
  }
#else
    // An output is still needed but it is not initialized.
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
#endif

  if (row_major_) {
    hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_RETURN_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
  }

  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                 &epilogue, sizeof(epilogue));

  // See
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true
  // with H100).
  size_t workspaceSize = (size_t)(1 << 25);  // suggested fixed value 32Mb
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulPreferenceCreate(&preference);
  hipblasLtMatmulPreferenceSetAttribute(preference,
                                       HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                       &workspaceSize, sizeof(workspaceSize));

  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  int returnedResults = 0;
  hipblasStatus_t cuda_status = hipblasLtMatmulAlgoGetHeuristic(
      cublasLt, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1,
      &heuristicResult, &returnedResults);
  ORT_ENFORCE(
      returnedResults > 0 && cuda_status == HIPBLAS_STATUS_SUCCESS,
      " Unable to find any suitable algorithm due to ",
      cublasGetErrorEnum(cuda_status), ", returnedResults=", returnedResults,
      ", alpha=", alpha_, ", beta=", beta_, ", n_inputs=", n_inputs,
      ", A_type=", CudaDataTypeToString(a_cuda_type),
      ", B_type=", CudaDataTypeToString(b_cuda_type),
      ", C_type=", CudaDataTypeToString(bias_cuda_type),
      ", result_type=", CudaDataTypeToString(d_cuda_type),
      ", bias_type=", CudaDataTypeToString(bias_cuda_type),
      ", scale_type=", CudaDataTypeToString(scale_cuda_type),
      ", computeType=", CublasComputeTypeToString(compute_type_),
      ", epilogue=", epilogue, ", smCount=", sm_count_, ", transA=", trans_A,
      ", transB=", trans_B,
      ", fastAccumulationMode=", (fast_accumulation_mode_ ? 1 : 0),
      ", shape_A=", shape_A[0], "x", shape_A[1], ", shape_B=", shape_B[0], "x",
      shape_B[1], ", shape_C=", (shape_C.NumDimensions() > 0 ? shape_C[0] : 0), "x",
      (shape_C.NumDimensions() > 1 ? shape_C[1] : 0), ", M=", M, ", N=", N, ", K=", K,
      ", lda=", lda, ", ldb=", ldb, ", ldd=", ldd,
      ", workspaceSize=", workspaceSize, ", rowMajor=", (row_major_ ? 1 : 0),
      ". Check NVIDIA documentation to see what combination is valid: ",
      "https://docs.nvidia.com/cuda/cublas/"
      "index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#"
      "cublasltmatmulalgogetheuristic.");

  void* workspace = nullptr;
  if (workspaceSize > 0) {
    CUDA_RETURN_IF_ERROR(hipMalloc((void**)&workspace, workspaceSize));
  }
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmul#cublasltmatmul
  const void* bias = has_bias ? p_input_c : p_output_y;
  cuda_status = hipblasLtMatmul(
      cublasLt, operationDesc, static_cast<const void*>(&alpha_), /* alpha */
      p_input_a,                                                  /* A */
      Adesc, p_input_b,                                           /* B */
      Bdesc, static_cast<const void*>(&beta_),                    /* beta */
      bias,                                                       /* C */
      Cdesc, p_output_y,                                          /* Y */
      Ddesc, &heuristicResult.algo,                               /* algo */
      workspace,                                                  /* workspace */
      workspaceSize, stream);                                     /* stream */
  ORT_ENFORCE(
      cuda_status == HIPBLAS_STATUS_SUCCESS,
      " Unable to run hipblasLtMatmul due to ", cublasGetErrorEnum(cuda_status),
      ", returnedResults=", returnedResults, ", alpha=", alpha_,
      ", n_inputs=", n_inputs, ", A_type=", CudaDataTypeToString(a_cuda_type),
      ", B_type=", CudaDataTypeToString(b_cuda_type),
      ", result_type=", CudaDataTypeToString(d_cuda_type),
      ", bias_type=", CudaDataTypeToString(bias_cuda_type),
      ", scale_type=", CudaDataTypeToString(scale_cuda_type),
      ", computeType=", CublasComputeTypeToString(compute_type_),
      ", epilogue=", epilogue, ", smCount=", sm_count_, ", transA=", trans_A,
      ", transB=", trans_B,
      ", fastAccumulationMode=", (fast_accumulation_mode_ ? 1 : 0),
      ", shape_A=", shape_A[0], "x", shape_A[1], ", shape_B=", shape_B[0], "x",
      shape_B[1], ", M=", M, ", N=", N, ", K=", K, ", lda=", lda, ", ldb=", ldb,
      ", ldd=", ldd, ", workspaceSize=", workspaceSize,
      ", rowMajor=", (row_major_ ? 1 : 0), ".");

  if (workspaceSize > 0) {
    CUDA_RETURN_IF_ERROR(hipFree(workspace));
  }

  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulPreferenceDestroy(preference));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Ddesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Cdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Bdesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatrixLayoutDestroy(Adesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtMatmulDescDestroy(operationDesc));
  CUBLAS_RETURN_IF_ERROR(hipblasLtDestroy(cublasLt));
  return Status::OK();
}

}  // namespace cuda
}  // namespace contrib
}  // namespace onnxruntime
