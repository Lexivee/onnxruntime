#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "quantize_linear.cuh"

#include <limits>

#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename InT, typename OutT>
struct Round;

template <>
struct Round<float, int8_t> {
  __device__ __forceinline__ int8_t operator()(float v, float scale, int8_t zero_point) const {
    int value = __float2int_rn(v / scale) + zero_point;
    return static_cast<int8_t>(max(std::numeric_limits<int8_t>::min(), min(std::numeric_limits<int8_t>::max(), value)));
  }
};

template <>
struct Round<float, uint8_t> {
  __device__ __forceinline__ uint8_t operator()(float v, float scale, uint8_t zero_point) const {
    int value = __float2int_rn(v / scale) + zero_point;
    return static_cast<uint8_t>(max(std::numeric_limits<uint8_t>::min(), min(std::numeric_limits<uint8_t>::max(), value)));
  }
};

template <>
struct Round<float, Float8E4M3FN> {
  __device__ __forceinline__ Float8E4M3FN operator()(float v, float scale, Float8E4M3FN zero_point) const {
    return Float8E4M3FN(v / scale);
  }
};

template <>
struct Round<float, Float8E4M3FNUZ> {
  __device__ __forceinline__ Float8E4M3FNUZ operator()(float v, float scale, Float8E4M3FNUZ zero_point) const {
    return Float8E4M3FNUZ(v / scale);
  }
};

template <>
struct Round<float, Float8E5M2> {
  __device__ __forceinline__ Float8E5M2 operator()(float v, float scale, Float8E5M2 zero_point) const {
    return Float8E5M2(v / scale);
  }
};

template <>
struct Round<float, Float8E5M2FNUZ> {
  __device__ __forceinline__ Float8E5M2FNUZ operator()(float v, float scale, Float8E5M2FNUZ zero_point) const {
    return Float8E5M2FNUZ(v / scale);
  }
};

template <>
struct Round<half, int8_t> {
  __device__ __forceinline__ int8_t operator()(half v, half scale, int8_t zero_point) const {
    int value = __half2int_rn(v / scale) + zero_point;
    return static_cast<int8_t>(max(std::numeric_limits<int8_t>::min(), min(std::numeric_limits<int8_t>::max(), value)));
  }
};

template <>
struct Round<half, uint8_t> {
  __device__ __forceinline__ int8_t operator()(half v, half scale, uint8_t zero_point) const {
    int value = __half2int_rn(v / scale) + zero_point;
    return static_cast<uint8_t>(max(std::numeric_limits<uint8_t>::min(), min(std::numeric_limits<uint8_t>::max(), value)));
  }
};

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void QuantizeLinearKernel(const InT* input, OutT* output, const InT* scale_ptr, const OutT* zero_point_ptr, CUDA_LONG N, Round<InT, OutT> round) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  InT scale = *scale_ptr;
  OutT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : static_cast<OutT>(0);
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = round(input[id], scale, zero_point);
      id += NumThreadsPerBlock;
    }
  }
}

template <class OutT, class InT>
Status CudaQuantizeLinear(hipStream_t stream, const InT* input, OutT* output, const InT* scale, const OutT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  QuantizeLinearKernel<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element),
      Round<InT, OutT>());
  return Status::OK();
}

template <class InT, class OutT, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void DequantizeLinearKernel(const InT* input, OutT* output, const OutT* scale_ptr, const InT* zero_point_ptr, CUDA_LONG N) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  OutT scale = *scale_ptr;
  InT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : static_cast<InT>(0);
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = static_cast<OutT>(input[id] - zero_point) * scale;
      id += NumThreadsPerBlock;
    }
  }
}

template <class InT, class OutT>
Status CudaDequantizeLinear(hipStream_t stream, const InT* input, OutT* output, const OutT* scale, const InT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  DequantizeLinearKernel<InT, OutT, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element));
  return Status::OK();
}

template Status CudaQuantizeLinear<int8_t, float>(hipStream_t stream, const float* input, int8_t* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, float>(hipStream_t stream, const float* input, uint8_t* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<int8_t, half>(hipStream_t stream, const half* input, int8_t* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, half>(hipStream_t stream, const half* input, uint8_t* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<Float8E4M3FN, float>(hipStream_t stream, const float* input, Float8E4M3FN* output, const float* scale, const Float8E4M3FN* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<Float8E4M3FNUZ, float>(hipStream_t stream, const float* input, Float8E4M3FNUZ* output, const float* scale, const Float8E4M3FNUZ* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<Float8E5M2, float>(hipStream_t stream, const float* input, Float8E5M2* output, const float* scale, const Float8E5M2* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<Float8E5M2FNUZ, float>(hipStream_t stream, const float* input, Float8E5M2FNUZ* output, const float* scale, const Float8E5M2FNUZ* zero_point, size_t num_of_element);

template Status CudaDequantizeLinear<int8_t, float>(hipStream_t stream, const int8_t* input, float* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, float>(hipStream_t stream, const uint8_t* input, float* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<int8_t, half>(hipStream_t stream, const int8_t* input, half* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, half>(hipStream_t stream, const uint8_t* input, half* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<Float8E4M3FN, float>(hipStream_t stream, const Float8E4M3FN* input, float* output, const float* scale, const Float8E4M3FN* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<Float8E4M3FNUZ, float>(hipStream_t stream, const Float8E4M3FNUZ* input, float* output, const float* scale, const Float8E4M3FNUZ* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<Float8E5M2, float>(hipStream_t stream, const Float8E5M2* input, float* output, const float* scale, const Float8E5M2* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<Float8E5M2FNUZ, float>(hipStream_t stream, const Float8E5M2FNUZ* input, float* output, const float* scale, const Float8E5M2FNUZ* zero_point, size_t num_of_element);

}  // namespace cuda
}  // namespace onnxruntime
