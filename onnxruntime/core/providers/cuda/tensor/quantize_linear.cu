#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "quantize_linear.cuh"

#include <limits>

#include "core/providers/cuda/cu_inc/common.cuh"

namespace onnxruntime {
namespace cuda {

template <typename InT, typename OutT>
struct Round;

template <>
struct Round<float, int8_t> {
  __device__ __forceinline__ int8_t operator()(float v, float scale, int8_t zero_point) const {
    int value = __float2int_rn(v / scale) + zero_point;
    return static_cast<int8_t>(max(std::numeric_limits<int8_t>::min(), min(std::numeric_limits<int8_t>::max(), value)));
  }
};

template <>
struct Round<float, uint8_t> {
  __device__ __forceinline__ uint8_t operator()(float v, float scale, uint8_t zero_point) const {
    int value = __float2int_rn(v / scale) + zero_point;
    return static_cast<uint8_t>(max(std::numeric_limits<uint8_t>::min(), min(std::numeric_limits<uint8_t>::max(), value)));
  }
};

template <>
struct Round<float, FloatE4M3> {
  __device__ __forceinline__ FloatE4M3 operator()(float v, float scale, FloatE4M3 zero_point) const {
    return FloatE4M3(v / scale);
  }
};

template <>
struct Round<float, FloatE5M2> {
  __device__ __forceinline__ FloatE5M2 operator()(float v, float scale, FloatE5M2 zero_point) const {
    return FloatE5M2(v / scale);
  }
};

template <>
struct Round<half, int8_t> {
  __device__ __forceinline__ int8_t operator()(half v, half scale, int8_t zero_point) const {
    int value = __half2int_rn(v / scale) + zero_point;
    return static_cast<int8_t>(max(std::numeric_limits<int8_t>::min(), min(std::numeric_limits<int8_t>::max(), value)));
  }
};

template <>
struct Round<half, uint8_t> {
  __device__ __forceinline__ int8_t operator()(half v, half scale, uint8_t zero_point) const {
    int value = __half2int_rn(v / scale) + zero_point;
    return static_cast<uint8_t>(max(std::numeric_limits<uint8_t>::min(), min(std::numeric_limits<uint8_t>::max(), value)));
  }
};

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void QuantizeLinearKernel(const InT* input, OutT* output, const InT* scale_ptr, const OutT* zero_point_ptr, CUDA_LONG N, Round<InT, OutT> round) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  InT scale = *scale_ptr;
  OutT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : static_cast<OutT>(0);
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = round(input[id], scale, zero_point);
      id += NumThreadsPerBlock;
    }
  }
}

template <class OutT, class InT>
Status CudaQuantizeLinear(hipStream_t stream, const InT* input, OutT* output, const InT* scale, const OutT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  QuantizeLinearKernel<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element),
      Round<InT, OutT>());
  return Status::OK();
}

template <class InT, class OutT, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void DequantizeLinearKernel(const InT* input, OutT* output, const OutT* scale_ptr, const InT* zero_point_ptr, CUDA_LONG N) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

  OutT scale = *scale_ptr;
  InT zero_point = zero_point_ptr != nullptr ? *zero_point_ptr : static_cast<InT>(0);
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = static_cast<OutT>(input[id] - zero_point) * scale;
      id += NumThreadsPerBlock;
    }
  }
}

template <class InT, class OutT>
Status CudaDequantizeLinear(hipStream_t stream, const InT* input, OutT* output, const OutT* scale, const InT* zero_point, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  DequantizeLinearKernel<InT, OutT, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      scale,
      zero_point,
      static_cast<int>(num_of_element));
  return Status::OK();
}

template Status CudaQuantizeLinear<int8_t, float>(hipStream_t stream, const float* input, int8_t* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, float>(hipStream_t stream, const float* input, uint8_t* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<int8_t, half>(hipStream_t stream, const half* input, int8_t* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<uint8_t, half>(hipStream_t stream, const half* input, uint8_t* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<FloatE4M3, float>(hipStream_t stream, const float* input, FloatE4M3* output, const float* scale, const FloatE4M3* zero_point, size_t num_of_element);
template Status CudaQuantizeLinear<FloatE5M2, float>(hipStream_t stream, const float* input, FloatE5M2* output, const float* scale, const FloatE5M2* zero_point, size_t num_of_element);

template Status CudaDequantizeLinear<int8_t, float>(hipStream_t stream, const int8_t* input, float* output, const float* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, float>(hipStream_t stream, const uint8_t* input, float* output, const float* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<int8_t, half>(hipStream_t stream, const int8_t* input, half* output, const half* scale, const int8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<uint8_t, half>(hipStream_t stream, const uint8_t* input, half* output, const half* scale, const uint8_t* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<FloatE4M3, float>(hipStream_t stream, const FloatE4M3* input, float* output, const float* scale, const FloatE4M3* zero_point, size_t num_of_element);
template Status CudaDequantizeLinear<FloatE5M2, float>(hipStream_t stream, const FloatE5M2* input, float* output, const float* scale, const FloatE5M2* zero_point, size_t num_of_element);

}  // namespace cuda
}  // namespace onnxruntime
