#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "transpose_impl.h"

namespace onnxruntime {
namespace cuda {

constexpr unsigned int TILE_DIM = 16;

template <typename T>
__global__ void Transpose3DKernel(const TArray<int64_t> input_shape,
                                  const TArray<int64_t> input_strides,
                                  const T* input_data, T* output_data) {
  __shared__ T tile[TILE_DIM * (TILE_DIM + 1)];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;

  tile[threadIdx.y * TILE_DIM + threadIdx.x] = input_data[blockIdx.z * input_strides[0] + y * input_shape[2] + x];
  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  output_data[blockIdx.z * input_strides[0] + y * input_shape[1] + x] = tile[threadIdx.x * TILE_DIM + threadIdx.y];
}

bool CanDoTranspose3D(int32_t rank,
                      const std::vector<int64_t>& input_dims,
                      const std::vector<size_t>& permutations) {
  if (rank == 3 &&
      // permutation is done in the last two dimensions.
      permutations[rank - 2] == (rank - 1) && permutations[rank - 1] == (rank - 2) &&
      // the last two dimensions are aligned with TILE_DIM.
      input_dims[rank - 2] % TILE_DIM == 0 && input_dims[rank - 1] % TILE_DIM == 0) {
    return true;
  }
  return false;
}

Status Transpose3DImpl(hipStream_t stream, size_t element_size,
                       const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
                       const void* input_data, void* output_data, int64_t N) {
  dim3 block_size(TILE_DIM, TILE_DIM);
  dim3 grid_size(static_cast<unsigned int>(input_shape[2] / TILE_DIM),
                 static_cast<unsigned int>(input_shape[1] / TILE_DIM),
                 static_cast<unsigned int>(input_shape[0]));

  switch (element_size) {
    case sizeof(int8_t):
      Transpose3DKernel<int8_t><<<grid_size, block_size, 0, stream>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data));
      break;
    case sizeof(int16_t):
      Transpose3DKernel<int16_t><<<grid_size, block_size, 0, stream>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data));
      break;
    case sizeof(int32_t):
      Transpose3DKernel<int32_t><<<grid_size, block_size, 0, stream>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data));
      break;
    case sizeof(int64_t):
      Transpose3DKernel<int64_t><<<grid_size, block_size, 0, stream>>>(
          input_shape, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data));
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

template <int element_size>
__global__ void Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim(
    const TArray<int64_t> input_strides, const void* input_data,
    const TArray<int64_t> output_strides, void* output_data,
    int64_t input_shape_2, CUDA_LONG N) {
  // coordinates will be: [d0, d1, d2, d3]
  CUDA_LONG d0 = blockIdx.z;
  CUDA_LONG d1 = blockIdx.y;
  CUDA_LONG d2 = threadIdx.y + blockIdx.x * blockDim.y;
  CUDA_LONG d3 = threadIdx.x;

  CUDA_LONG input_index = (d0 * input_strides[0] +
                           d1 * input_strides[1] +
                           d2 * input_strides[2]) /
                              (4 * sizeof(int) / element_size) +
                          d3 * input_strides[3];

  CUDA_LONG output_index = (d0 * output_strides[0] +
                            d1 * output_strides[1] +
                            d2 * output_strides[2]) /
                               (4 * sizeof(int) / element_size) +
                           d3 * output_strides[3];

  const int4* v_input = reinterpret_cast<const int4*>(input_data);
  int4* v_output = reinterpret_cast<int4*>(output_data);

  if (input_index < N && output_index < N && d2 < input_shape_2) {
    v_output[output_index] = v_input[input_index];
  }
}

bool CanDoTranspose4DParallelizeMultipleElementsPerThreadInInnermostDim(const hipDeviceProp_t& prop,
                                                                        size_t element_size,
                                                                        int32_t rank,
                                                                        const std::vector<int64_t>& input_dims,
                                                                        const std::vector<size_t>& permutations) {
  if (rank == 4 &&
      // the permutations is not on the last dimension.
      permutations[3] == 3) {
    unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.

    if (input_dims[3] / num_elements_per_thread <= prop.maxThreadsPerBlock &&
        ((input_dims[3] % num_elements_per_thread) == 0)) {
      return true;
    }
  }
  return false;
}

Status Transpose4DParallelizeMultipleElementsPerThreadInInnermostDim(
    const hipDeviceProp_t& prop, hipStream_t stream, size_t element_size,
    const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
    const void* input_data, const TArray<int64_t>& output_strides,
    void* output_data, int N) {
  unsigned int num_elements_per_thread = 4 * sizeof(int) / static_cast<unsigned int>(element_size);  // int4 is used in the kernel to access data.
  // There are 2 constrains when luanching the kernels
  // 1. block_size_x * block_size_y <= prop.maxThreadsPerBlock
  // 2. block_size_y * num_block_ext >= input_shape[2]
  int64_t block_size_x = input_shape[3] / num_elements_per_thread;
  int64_t max_block_size_y = prop.maxThreadsPerBlock / block_size_x;
  int64_t block_size_y = min(input_shape[2], max_block_size_y);
  int64_t num_block_ext = CeilDiv(input_shape[2], block_size_y);

  dim3 block_size(static_cast<unsigned int>(block_size_x), static_cast<unsigned int>(block_size_y));
  dim3 grid_size(static_cast<unsigned int>(num_block_ext),
                 static_cast<unsigned int>(input_shape[1]),
                 static_cast<unsigned int>(input_shape[0]));

  std::cout << "Transpose4DParallelizeMultipleElementsPerThreadInInnermostDim\n";
  std::cout << "shape: [" << input_shape[0] << "," << input_shape[1] << "," << input_shape[2] << "," << input_shape[3] << "]\n";
  std::cout << "block_size.x: " << block_size.x << " block_size.y: " << block_size.y
            << " grid_size.x: " << grid_size.x << " grid_size.y: " << grid_size.y << " grid_size.z: " << grid_size.z << "\n";

  switch (element_size) {
    case sizeof(int8_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int8_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int16_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int16_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int32_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int32_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    case sizeof(int64_t):
      Transpose4DKernelParallelizeMultipleElementsPerThreadInInnermostDim<sizeof(int64_t)>
          <<<grid_size, block_size, 0, stream>>>(
              input_strides, input_data,
              output_strides, output_data,
              input_shape[2],
              N / num_elements_per_thread);
      break;
    default:
      // User will not hit this as this kernel is for fixed element size tensors only
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

__global__ void Transpose4DKernelParallelizeOneElementPerThread(
    const TArray<int64_t> input_strides, const int8_t* input_data,
    const TArray<int64_t> output_strides, int8_t* output_data,
    size_t element_size, int64_t input_shape_2, CUDA_LONG N) {
  // coordinates will be: [d0, d1, d2, d3]
  CUDA_LONG d0 = blockIdx.z;
  CUDA_LONG d1 = blockIdx.y;
  CUDA_LONG d2 = threadIdx.y + blockIdx.x * blockDim.y;
  CUDA_LONG d3 = threadIdx.x;

  CUDA_LONG input_index = d0 * input_strides[0] +
                          d1 * input_strides[1] +
                          d2 * input_strides[2] +
                          d3 * input_strides[3];

  CUDA_LONG output_index = d0 * output_strides[0] +
                           d1 * output_strides[1] +
                           d2 * output_strides[2] +
                           d3 * output_strides[3];

  if (input_index < N && output_index < N && d2 < input_shape_2) {
    const int8_t* input_data_to_be_copied = input_data + (input_index * element_size);
    int8_t* output_data_to_be_copied = output_data + (output_index * element_size);

    // copy over the bytes
    for (size_t iter = 0; iter < element_size; ++iter) {
      *output_data_to_be_copied++ = *input_data_to_be_copied++;
    }
  }
}

bool CanDoTranspose4DParallelizeOneElementPerThread(const hipDeviceProp_t& prop,
                                                    size_t element_size,
                                                    int32_t rank,
                                                    const std::vector<int64_t>& input_dims,
                                                    const std::vector<size_t>& permutations) {
  if (rank == 4) {
    if (input_dims[3] <= prop.maxThreadsPerBlock) {
      return true;
    }
  }
  return false;
}

Status Transpose4DParallelizeOneElementPerThread(
    const hipDeviceProp_t& prop, hipStream_t stream, size_t element_size,
    const TArray<int64_t>& input_shape, const TArray<int64_t>& input_strides,
    const void* input_data, const TArray<int64_t>& output_strides,
    void* output_data, int N) {
  if (element_size != sizeof(int8_t) &&
      element_size != sizeof(int16_t) &&
      element_size != sizeof(int32_t) &&
      element_size != sizeof(int64_t)) {
    // User will not hit this as this kernel is for fixed element size tensors only
    return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                           element_size);
  }
  
  // There are 2 constrains when luanching the kernels
  // 1. block_size_x * block_size_y <= prop.maxThreadsPerBlock
  // 2. block_size_y * num_block_ext >= input_shape[2]
  int64_t block_size_x = input_shape[3];
  int64_t max_block_size_y = prop.maxThreadsPerBlock / block_size_x;
  int64_t block_size_y = min(input_shape[2], max_block_size_y);
  int64_t num_block_ext = CeilDiv(input_shape[2], block_size_y);

  dim3 block_size(static_cast<unsigned int>(block_size_x), static_cast<unsigned int>(block_size_y));
  dim3 grid_size(static_cast<unsigned int>(num_block_ext),
                 static_cast<unsigned int>(input_shape[1]),
                 static_cast<unsigned int>(input_shape[0]));

  std::cout << "Transpose4DKernelParallelizeOneElementPerThread\n";
  std::cout << "shape: [" << input_shape[0] << "," << input_shape[1] << "," << input_shape[2] << "," << input_shape[3] << "]\n";
  std::cout << "block_size.x: " << block_size.x << " block_size.y: " << block_size.y
            << " grid_size.x: " << grid_size.x << " grid_size.y: " << grid_size.y << " grid_size.z: " << grid_size.z << "\n";

  Transpose4DKernelParallelizeOneElementPerThread<<<grid_size, block_size, 0, stream>>>(
      input_strides, reinterpret_cast<const int8_t*>(input_data),
      output_strides, reinterpret_cast<int8_t*>(output_data),
      element_size, input_shape[2], N);

  return Status::OK();
}

template <typename T>
__global__ void TransposeKernel(int32_t shape_rank, const TArray<int64_t> input_strides,
                                const T* input_data, const TArray<fast_divmod> output_strides, T* output_data, CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  CUDA_LONG input_index = 0;
  CUDA_LONG output_index = id;

#pragma unroll
  for (auto dim = 0; dim < input_strides.Capacity(); ++dim) {
    if (dim >= shape_rank) {
      break;
    }
    int out_coord, r;
    output_strides[dim].divmod(output_index, out_coord, r);
    output_index = r;
    input_index += input_strides[dim] * out_coord;
  }
  output_data[id] = input_data[input_index];
}

Status TransposeImpl(hipStream_t stream, size_t element_size, int32_t shape_rank, const TArray<int64_t>& input_strides,
                     const void* input_data, const TArray<fast_divmod>& fdm_output_strides, void* output_data, int N) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));
  switch (element_size) {
    case sizeof(int8_t):
      TransposeKernel<int8_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int8_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int8_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int16_t):
      TransposeKernel<int16_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int16_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int16_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int32_t):
      TransposeKernel<int32_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int32_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int32_t>::MappedType*>(output_data),
          N);
      break;
    case sizeof(int64_t):
      TransposeKernel<int64_t><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
          shape_rank, input_strides,
          reinterpret_cast<const ToCudaType<int64_t>::MappedType*>(input_data),
          fdm_output_strides,
          reinterpret_cast<ToCudaType<int64_t>::MappedType*>(output_data),
          N);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for transpose on CUDA. Element size was ",
                             element_size);
  }

  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
