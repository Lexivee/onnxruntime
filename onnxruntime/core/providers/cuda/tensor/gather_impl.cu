#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "gather_impl.h"

namespace onnxruntime {
namespace cuda {

template <typename T, typename Tin, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _GatherKernel(
    const int64_t input_block_size,
    const int64_t indices_max,
    const Tin* indices_data,
    const fast_divmod* div_strides,
    const T* input_data,
    T* output_data,
    const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N, NumElementsPerThread);

  #pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      CUDA_LONG input_index = 0;
      int input_block_index, block_offset;
      div_strides[0].divmod(id, input_block_index, block_offset);
      int indices_index, offset;
      div_strides[1].divmod(block_offset, indices_index, offset);
      int block_size = div_strides[1].d_;
      int64_t idx = indices_data[indices_index];
      if (idx < 0 || idx >= indices_max) {
        output_data[id] = 0;
        return;
      }

      input_index = input_block_index * input_block_size + idx * block_size + offset;
      output_data[id] = input_data[input_index];
      id += NumThreadsPerBlock;
    }
  }
}

template <typename T, typename Tin>
void GatherImpl(
    const int64_t input_block_size,
    const int64_t indices_max,
    const Tin* indices_data,
    const fast_divmod* div_strides,
    const T* input_data,
    T* output_data,
    const size_t N) {
  int blocksPerGrid = static_cast<int>(CeilDiv(N, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  _GatherKernel<T, Tin, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread>\
    <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      input_block_size, indices_max, indices_data, div_strides, input_data, output_data, (CUDA_LONG)N);
}

#define SPECIALIZED_IMPL(T)                                                                                                                                                                                          \
  template void GatherImpl<T, int32_t>(const int64_t input_block_size, const int64_t indices_max, const int32_t* indices_data, const fast_divmod* div_strides, const T* input_data, T* output_data, const size_t N); \
  template void GatherImpl<T, int64_t>(const int64_t input_block_size, const int64_t indices_max, const int64_t* indices_data, const fast_divmod* div_strides, const T* input_data, T* output_data, const size_t N);

SPECIALIZED_IMPL(int8_t)
SPECIALIZED_IMPL(int16_t)
SPECIALIZED_IMPL(int32_t)
SPECIALIZED_IMPL(int64_t)
SPECIALIZED_IMPL(uint8_t)
SPECIALIZED_IMPL(uint16_t)
SPECIALIZED_IMPL(uint32_t)
SPECIALIZED_IMPL(uint64_t)
SPECIALIZED_IMPL(half)
SPECIALIZED_IMPL(float)
SPECIALIZED_IMPL(double)
SPECIALIZED_IMPL(bool)

template <typename T, typename Tin, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _GatherGradKernel(
    const int64_t input_block_size,
    const int64_t indices_max,
    const Tin* indices_data,
    const fast_divmod* div_strides,
    const T* grad_data,
    T* output_data,
    const CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N, NumElementsPerThread);

  #pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      CUDA_LONG input_index = 0;
      int input_block_index, block_offset;
      div_strides[0].divmod(id, input_block_index, block_offset);
      int indices_index, offset;
      div_strides[1].divmod(block_offset, indices_index, offset);
      int block_size = div_strides[1].d_;
      int64_t idx = indices_data[indices_index];
      if (idx < 0 || idx >= indices_max) {
        output_data[id] = 0;
        return;
      }

      input_index = input_block_index * input_block_size + idx * block_size + offset;
      atomicAdd(output_data + input_index, grad_data[id]);
      id += NumThreadsPerBlock;
    }
  }
}

template <typename T, typename Tin>
void GatherGradImpl(
    const int64_t input_block_size,
    const int64_t indices_max,
    const Tin* indices_data,
    const fast_divmod* div_strides,
    const T* grad_data,
    T* output_data,
    const size_t N) {
  int blocksPerGrid = static_cast<int>(CeilDiv(N, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  _GatherGradKernel<T, Tin, GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread>\
    <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      input_block_size, indices_max, indices_data, div_strides, grad_data, output_data, (CUDA_LONG)N);
}

#define SPECIALIZED_GRAD_IMPL(T)                                                                                                                                                                                        \
  template void GatherGradImpl<T, int32_t>(const int64_t input_block_size, const int64_t indices_max, const int32_t* indices_data, const fast_divmod* div_strides, const T* grad_data, T* output_data, const size_t N); \
  template void GatherGradImpl<T, int64_t>(const int64_t input_block_size, const int64_t indices_max, const int64_t* indices_data, const fast_divmod* div_strides, const T* grad_data, T* output_data, const size_t N);

// TODO: AtomicAdd doesn't have full support in all date types

//SPECIALIZED_GRAD_IMPL(int8_t)
//SPECIALIZED_GRAD_IMPL(int16_t)
SPECIALIZED_GRAD_IMPL(int32_t)
//SPECIALIZED_GRAD_IMPL(int64_t)
//SPECIALIZED_GRAD_IMPL(uint8_t)
//SPECIALIZED_GRAD_IMPL(uint16_t)
SPECIALIZED_GRAD_IMPL(uint32_t)
//SPECIALIZED_GRAD_IMPL(uint64_t)
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 700
SPECIALIZED_GRAD_IMPL(half)
#endif
SPECIALIZED_GRAD_IMPL(float)
//SPECIALIZED_GRAD_IMPL(double)
//SPECIALIZED_GRAD_IMPL(bool)

}  // namespace cuda
}  // namespace onnxruntime
