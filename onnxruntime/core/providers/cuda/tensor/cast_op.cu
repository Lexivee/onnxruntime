#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "cast_op.cuh"

#include <limits>

#include "core/providers/cuda/cu_inc/common.cuh"

#if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
#include "hip/hip_fp8.h"
#endif

namespace onnxruntime {
namespace cuda {

template <typename OutT, typename InT>
struct Cast;

template <>
struct Cast<Float8E4M3FN, float> {
  __device__ __forceinline__ Float8E4M3FN operator()(float v) const {
    return Float8E4M3FN(__hip_cvt_float_to_fp8(v, __HIP_NOSAT, __HIP_E4M3_FNUZ));
  }
};

template <>
struct Cast<float, Float8E4M3FN> {
  __device__ __forceinline__ float operator()(Float8E4M3FN v) const {
    return __half2float(__hip_cvt_fp8_to_halfraw(v.val, __HIP_E4M3_FNUZ));
  }
};

template <int NumThreadsPerBlock, int NumElementsPerThread, typename OutT, typename InT>
__global__ void CastKernel(const InT* input, OutT* output, CUDA_LONG N, Cast<OutT, InT> cast) {
  CUDA_LONG id = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;

#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      output[id] = cast(input[id]);
      id += NumThreadsPerBlock;
    }
  }
}
template <class OutT, class InT>
Status CudaCast(hipStream_t stream, const InT* input, OutT* output, size_t num_of_element) {
  if (num_of_element <= 0)
    return Status::OK();

  int blocksPerGrid = static_cast<int>(CeilDiv(num_of_element, GridDim::maxThreadsPerBlock * GridDim::maxElementsPerThread));
  CastKernel<GridDim::maxThreadsPerBlock, GridDim::maxElementsPerThread><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(
      input,
      output,
      static_cast<int>(num_of_element),
      Cast<OutT, InT>());
  return Status::OK();
}

template Status CudaCast<Float8E4M3FN, float>(hipStream_t stream, const float* input, Float8E4M3FN* output, size_t num_of_element);
template Status CudaCast<float, Float8E4M3FN>(hipStream_t stream, const Float8E4M3FN* input, float* output, size_t num_of_element);

}  // namespace cuda
}  // namespace onnxruntime
