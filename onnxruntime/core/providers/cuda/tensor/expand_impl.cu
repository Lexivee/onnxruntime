#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "expand_impl.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void ExpandKernel(
    const int32_t rank,
    const size_t N,
    const size_t N_input,
    const T* input_data,
    T* output_data,
    const TArray<fast_divmod> fdm_input_dims,
    const TArray<fast_divmod> fdm_output_dims,
    const TArray<fast_divmod> fdm_output_subdim_size) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  // initialize
  auto output_index = id;
  auto input_index = 0;
  auto input_subdim_size = N_input;
  auto out_coord = output_index;
  // use striding when tensor is larger than grid
  int stride = blockDim.x * gridDim.x;

  // translate indices to coordinates. copy expanded dims from source
  while (output_index < N) {
    for (auto i = 0; i < rank; i++) {
      input_subdim_size = fdm_input_dims.data_[i].div(input_subdim_size);
      auto new_out_coord = fdm_output_subdim_size.data_[i].div(out_coord);
      auto in_coord = (new_out_coord > (fdm_input_dims.data_[i].d_ - 1)) ? fdm_input_dims.data_[i].d_ - 1 : new_out_coord;
      input_index += input_subdim_size * in_coord;
      out_coord -= new_out_coord * fdm_output_subdim_size.data_[i].d_;
    }
    output_data[output_index] = input_data[input_index];
    output_index += stride;
    out_coord = output_index;
    input_subdim_size = N_input;
    input_index = 0;
  }
}

Status ExpandImpl(
    const size_t element_size,
    const int32_t rank,
    const size_t N,
    const size_t N_input,
    const void* input_data,
    void* output_data,
    const TArray<fast_divmod>* fdm_input_dims,
    const TArray<fast_divmod>* fdm_output_dims,
    const TArray<fast_divmod>* fdm_output_subdim_size) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(N) / GridDim::maxThreadsPerBlock));

  switch (element_size) {
    case sizeof(uint8_t):
      ExpandKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          rank, N, N_input,
          reinterpret_cast<const ToCudaType<uint8_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<uint8_t>::MappedType*>(output_data),
          *fdm_input_dims, *fdm_output_dims, *fdm_output_subdim_size);
      break;
    case sizeof(uint16_t):
      ExpandKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          rank, N, N_input,
          reinterpret_cast<const ToCudaType<uint16_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<uint16_t>::MappedType*>(output_data),
          *fdm_input_dims, *fdm_output_dims, *fdm_output_subdim_size);
      break;
    case sizeof(uint32_t):
      ExpandKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          rank, N, N_input,
          reinterpret_cast<const ToCudaType<uint32_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<uint32_t>::MappedType*>(output_data),
          *fdm_input_dims, *fdm_output_dims, *fdm_output_subdim_size);
      break;
    case sizeof(uint64_t):
      ExpandKernel<<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
          rank, N, N_input,
          reinterpret_cast<const ToCudaType<uint64_t>::MappedType*>(input_data),
          reinterpret_cast<ToCudaType<uint64_t>::MappedType*>(output_data),
          *fdm_input_dims, *fdm_output_dims, *fdm_output_subdim_size);
      break;
    default:
      return ORT_MAKE_STATUS(ONNXRUNTIME, FAIL, "Type not supported for Expand operator");
  }
  return Status::OK();
}

}  // namespace cuda
}  // namespace onnxruntime
