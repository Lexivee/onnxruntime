#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "optimizers.h"
#include "core/providers/cuda/cuda_common.h"


namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _SGDOptimizer(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  weights_out[id] = weights[id] - ((*eta) * gradients[id]);
}

template <typename T>
void SGDOptimizerImpl(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _SGDOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      weights,
      gradients,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL__SGDOptimizerImpl(T)      \
template void SGDOptimizerImpl(                    \
    const T* eta,                                  \
    const T* weights,                              \
    const T* gradients,                            \
    T* weights_out,                                \
    size_t count);

SPECIALIZED_IMPL__SGDOptimizerImpl(float)

template <typename T>
__global__ void _AdamOptimizer(
    const T* eta,
    const int64_t* update_count,
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    int64_t* update_count_out,
    CUDA_LONG N) {

  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  // Regularize gradient
  T g_regularized = lambda * weights[id] + grads[id];

  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] + ((1 - alpha) * g_regularized);

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] + ((1 - beta) * g_regularized * g_regularized);

  // Update learning rate - Use the updated eta for the final weight update
  const float numerator = _Sqrt(1 - _Pow(beta, static_cast<float>(*update_count)));
  const float denom = (1 - _Pow(alpha, static_cast<float>(*update_count)));
  const float eta_new = (*eta) * numerator / denom;

  weights_out[id] = weights[id] - ((eta_new * moment_1_out[id]) / (_Sqrt(moment_2_out[id]) + epsilon));
  *update_count_out = (*update_count) + 1;
}

template <typename T>
void AdamOptimizerImpl(
    const T* eta,
    const int64_t* update_count,
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    int64_t* update_count_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _AdamOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      update_count,
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      weights_out,
      moment_1_out,
      moment_2_out,
      update_count_out,
      N);
}

#define SPECIALIZED_IMPL__AdamOptimizerImpl(T)      \
template void AdamOptimizerImpl(                    \
    const T* eta,                                   \
    const int64_t* update_count,                    \
    const T* weights,                               \
    const T* grads,                                 \
    const T* moment_1,                              \
    const T* moment_2,                              \
    float alpha,                                    \
    float beta,                                     \
    float lambda,                                   \
    float epsilon,                                  \
    T* weights_out,                                 \
    T* moment_1_out,                                \
    T* moment_2_out,                                \
    int64_t* update_count_out,                      \
    size_t count);

SPECIALIZED_IMPL__AdamOptimizerImpl(float)

template <typename T1, typename T2, typename T3>
__global__ void _LambComputeDirection(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  const T3 one = T3(1.0);
  const T3 g = T3(grads[id]);

  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] + \
    (one - alpha) * g;

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] + \
    (one - beta) * g * g;

  // Save regularized update direction to output.
  update_direction[id] = lambda * weights[id] + \
    T1(moment_1_out[id] / (_Sqrt(moment_2_out[id]) + epsilon));
}

template <typename T1, typename T2, typename T3>
void LambComputeDirectionImpl(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    size_t count) {
  int blocksPerGrid = \
    (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambComputeDirection<T1, T2, T3>\
    <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      update_direction,
      moment_1_out,
      moment_2_out,
      N);
}

#define SPECIALIZED_IMPL_LambComputeDirectionImpl(T1, T2, T3) \
template void LambComputeDirectionImpl(                  \
    const T1* weights,                                   \
    const T2* grads,                                     \
    const T3* moment_1,                                  \
    const T3* moment_2,                                  \
    T3 alpha,                                            \
    T3 beta,                                             \
    T1 lambda,                                           \
    T3 epsilon,                                          \
    T2* weights_out,                                     \
    T3* moment_1_out,                                    \
    T3* moment_2_out,                                    \
    size_t count);

SPECIALIZED_IMPL_LambComputeDirectionImpl(float, float, float)
SPECIALIZED_IMPL_LambComputeDirectionImpl(double, double, double)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, half)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, float)

template <typename T1, typename T2>
__global__ void _LambUpdate(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T1* update_direction,
    T2* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Compute new weight using the saved update direction.
  weights_out[id] = weights[id] - \
    (*w_norm) / (*r_norm) * T2((*eta) * update_direction[id]);
}

template <typename T1, typename T2>
void LambUpdateImpl(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T1* update_direction,
    T2* weights_out,
    size_t count) {
  int blocksPerGrid = \
    (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambUpdate<T1, T2><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      r_norm,
      w_norm,
      weights,
      update_direction,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL_LambUpdate(T1, T2) \
template void LambUpdateImpl(               \
    const T1* eta,                          \
    const T2* r_norm,                       \
    const T2* w_norm,                       \
    const T2* weights,                      \
    const T1* update_direction,             \
    T2* weights_out,                        \
    size_t count);

SPECIALIZED_IMPL_LambUpdate(float, float)
SPECIALIZED_IMPL_LambUpdate(double, double)
SPECIALIZED_IMPL_LambUpdate(half, float)

template <typename T1, typename T2>
__global__ void _LambScalarL2NormReduction(
    const T1* value,
    T2* value_out) {
  *value_out = _Abs(*value);
}

template <typename T1, typename T2>
void LambScalarL2NormReductionImpl(
    const T1* value,
    T2* value_out) {
  _LambScalarL2NormReduction<T1, T2><<<1, 1, 0>>>(
      value,
      value_out);
}

#define SPECIALIZED_IMPL_LambScalarL2NormReduction(T1, T2) \
template void LambScalarL2NormReductionImpl(               \
    const T1* value,                                       \
    T2* value_out);

SPECIALIZED_IMPL_LambScalarL2NormReduction(float, float)
SPECIALIZED_IMPL_LambScalarL2NormReduction(double, double)
SPECIALIZED_IMPL_LambScalarL2NormReduction(half, float)

}  // namespace cuda
}  // namespace onnxruntime
