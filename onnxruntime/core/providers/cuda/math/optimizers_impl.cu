#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "optimizers.h"
#include "core/providers/cuda/cuda_common.h"


namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _SGDOptimizer(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  weights_out[id] = weights[id] - ((*eta) * gradients[id]);
}

template <typename T>
void SGDOptimizerImpl(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _SGDOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      weights,
      gradients,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL__SGDOptimizerImpl(T)      \
template void SGDOptimizerImpl(                    \
    const T* eta,                                  \
    const T* weights,                              \
    const T* gradients,                            \
    T* weights_out,                                \
    size_t count);

SPECIALIZED_IMPL__SGDOptimizerImpl(float)

template <typename T>
__global__ void _AdamOptimizer(
    const T* eta,
    const int64_t* update_count,
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    int64_t* update_count_out,
    CUDA_LONG N) {

  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);

  // Regularize gradient
  T g_regularized = lambda * weights[id] + grads[id];

  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] + ((1 - alpha) * g_regularized);

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] + ((1 - beta) * g_regularized * g_regularized);

  // Update learning rate - Use the updated eta for the final weight update
  float numerator = _Sqrt(1 - _Pow(beta, static_cast<float>(*update_count)));
  float denom = (1 - _Pow(alpha, static_cast<float>(*update_count)));
  float eta_new = (*eta) * numerator / denom;

  weights_out[id] = weights[id] - ((eta_new * moment_1_out[id]) / (_Sqrt(moment_2_out[id]) + epsilon));
  *update_count_out = (*update_count) + 1;
}

template <typename T>
void AdamOptimizerImpl(
    const T* eta,
    const int64_t* update_count,
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    int64_t* update_count_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _AdamOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      update_count,
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      weights_out,
      moment_1_out,
      moment_2_out,
      update_count_out,
      N);
}

#define SPECIALIZED_IMPL__AdamOptimizerImpl(T)      \
template void AdamOptimizerImpl(                    \
    const T* eta,                                   \
    const int64_t* update_count,                    \
    const T* weights,                               \
    const T* grads,                                 \
    const T* moment_1,                              \
    const T* moment_2,                              \
    float alpha,                                    \
    float beta,                                     \
    float lambda,                                   \
    float epsilon,                                  \
    T* weights_out,                                 \
    T* moment_1_out,                                \
    T* moment_2_out,                                \
    int64_t* update_count_out,                      \
    size_t count);

SPECIALIZED_IMPL__AdamOptimizerImpl(float)

template <typename T>
__global__ void _LambComputeDirection(
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] + (1 - alpha) * grads[id];

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] + (1 - beta) * grads[id] * grads[id];

  // Save regularized update direction to output.
  weights_out[id] = lambda * weights[id] + moment_1_out[id] / (_Sqrt(moment_2_out[id]) + epsilon);
}

template <typename T>
void LambComputeDirectionImpl(
    const T* weights,
    const T* grads,
    const T* moment_1,
    const T* moment_2,
    float alpha,
    float beta,
    float lambda,
    float epsilon,
    T* weights_out,
    T* moment_1_out,
    T* moment_2_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambComputeDirection<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      weights_out,
      moment_1_out,
      moment_2_out,
      N);
}

#define SPECIALIZED_IMPL_LambComputeDirectionImpl(T)\
template void LambComputeDirectionImpl(            \
    const T* weights,                               \
    const T* grads,                                 \
    const T* moment_1,                              \
    const T* moment_2,                              \
    float alpha,                                    \
    float beta,                                     \
    float lambda,                                   \
    float epsilon,                                  \
    T* weights_out,                                 \
    T* moment_1_out,                                \
    T* moment_2_out,                                \
    size_t count);

SPECIALIZED_IMPL_LambComputeDirectionImpl(float)

template <typename T>
__global__ void _LambUpdate(
    const T* eta,
    const T* r_norm,
    const T* w_norm,
    const T* weights,
    const T* update_direction,
    T* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Compute new weight using the saved update direction.
  weights_out[id] = weights[id] - (*eta) * (*w_norm) / (*r_norm) * update_direction[id];
}

template <typename T>
void LambUpdateImpl(
    const T* eta,
    const T* r_norm,
    const T* w_norm,
    const T* weights,
    const T* update_direction,
    T* weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambUpdate<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      r_norm,
      w_norm,
      weights,
      update_direction,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL_LambUpdate(T)     \
template void LambUpdateImpl(              \
    const T* eta,                          \
    const T* r_norm,                       \
    const T* w_norm,                       \
    const T* weights,                      \
    const T* update_direction,             \
    T* weights_out,                        \
    size_t count);

SPECIALIZED_IMPL_LambUpdate(float)

template <typename T>
__global__ void _LambScalarL2NormReduction(
    const T* value,
    T* value_out) {
  *value_out = _Abs(*value);
}

template <typename T>
void LambScalarL2NormReductionImpl(
    const T* value,
    T* value_out) {
  _LambScalarL2NormReduction<T><<<1, 1, 0>>>(
      value,
      value_out);
}

#define SPECIALIZED_IMPL_LambScalarL2NormReduction(T)     \
template void LambScalarL2NormReductionImpl(              \
    const T* value,                                       \
    T* value_out);

SPECIALIZED_IMPL_LambScalarL2NormReduction(float)
}  // namespace cuda
}  // namespace onnxruntime
