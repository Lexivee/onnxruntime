#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "optimizers.h"
#include "core/providers/cuda/cuda_common.h"

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _SGDOptimizer(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  weights_out[id] = weights[id] - ((*eta) * gradients[id]);
}

template <typename T>
void SGDOptimizerImpl(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _SGDOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      weights,
      gradients,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL__SGDOptimizerImpl(T) \
  template void SGDOptimizerImpl(             \
      const T* eta,                           \
      const T* weights,                       \
      const T* gradients,                     \
      T* weights_out,                         \
      size_t count);

SPECIALIZED_IMPL__SGDOptimizerImpl(float)

template <typename T1, typename T2, typename T3, typename T4, typename T_GRAD, bool update_fp16_weight>
__global__ void _AdamOptimizer(
    const T1* eta,
    const T2 update_count,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    T4 alpha,
    T4 beta,
    T4 lambda,
    T4 epsilon,
    T3* weights_out,
    T4* moment_1_out,
    T4* moment_2_out,
    half* fp16_weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Regularize gradient.
  const T4 g_regularized = lambda * T4(weights[id]) + T4(grads[id]);

  // A shared constant.
  const T4 one = T4(1.0f);

  // Compute exponentially-averaged historical gradient.
  T4 m1o = alpha * moment_1[id] + (one - alpha) * g_regularized;

  // Compute exponentially-averaged historical squared gradient.
  T4 m2o = beta * moment_2[id] + (one - beta) * g_regularized * g_regularized;

  // Update learning rate - Use the updated eta for the final weight update.
  const T4 count = T4(static_cast<long long>(update_count));
  const T4 numerator = _Sqrt(one - _Pow(beta, count));
  const T4 denom = one - _Pow(alpha, count);
  const T4 eta_new = T4(*eta) * numerator / denom;

  // Compute the new weight.
  weights_out[id] = weights[id] -
                    T3(eta_new * m1o / (_Sqrt(m2o) + epsilon));

  if (update_fp16_weight) {
    fp16_weights_out[id] = static_cast<half>(weights_out[id]);
  }

  moment_1_out[id] = m1o;
  moment_2_out[id] = m2o;
}

template <typename T1, typename T2, typename T3, typename T4, typename T_GRAD>
void AdamOptimizerImpl(
    const T1* eta,
    const T2 update_count,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    T4 alpha,
    T4 beta,
    T4 lambda,
    T4 epsilon,
    T3* weights_out,
    T4* moment_1_out,
    T4* moment_2_out,
    half* fp16_weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);

  if (fp16_weights_out != nullptr) {
    _AdamOptimizer<T1, T2, T3, T4, T_GRAD, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        eta,
        update_count,
        weights,
        grads,
        moment_1,
        moment_2,
        alpha,
        beta,
        lambda,
        epsilon,
        weights_out,
        moment_1_out,
        moment_2_out,
        fp16_weights_out,
        N);
  } else {
    _AdamOptimizer<T1, T2, T3, T4, T_GRAD, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        eta,
        update_count,
        weights,
        grads,
        moment_1,
        moment_2,
        alpha,
        beta,
        lambda,
        epsilon,
        weights_out,
        moment_1_out,
        moment_2_out,
        nullptr,
        N);
  }
}

#define SPECIALIZED_AdamOptimizerImpl(T1, T2, T3, T4, T_GRAD) \
  template void AdamOptimizerImpl(                            \
      const T1* eta,                                          \
      const T2 update_count,                                  \
      const T3* weights,                                      \
      const T_GRAD* grads,                                    \
      const T4* moment_1,                                     \
      const T4* moment_2,                                     \
      T4 alpha,                                               \
      T4 beta,                                                \
      T4 lambda,                                              \
      T4 epsilon,                                             \
      T3* weights_out,                                        \
      T4* moment_1_out,                                       \
      T4* moment_2_out,                                       \
      half* fp16_weights_out,                                 \
      size_t count);

SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, float)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, float)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, float)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, half)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, half)
template <typename T1, typename T2, typename T3>
__global__ void _LambComputeDirection(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  const T3 one = T3(1.0);
  const T3 g = T3(grads[id]);

  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] +
                     (one - alpha) * g;

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] +
                     (one - beta) * g * g;

  // Save regularized update direction to output.
  update_direction[id] = lambda * weights[id] +
                         T1(moment_1_out[id] / (_Sqrt(moment_2_out[id]) + epsilon));
}

template <typename T1, typename T2, typename T3>
void LambComputeDirectionImpl(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    size_t count) {
  int blocksPerGrid =
      (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambComputeDirection<T1, T2, T3><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      update_direction,
      moment_1_out,
      moment_2_out,
      N);
}

#define SPECIALIZED_IMPL_LambComputeDirectionImpl(T1, T2, T3) \
  template void LambComputeDirectionImpl(                     \
      const T1* weights,                                      \
      const T2* grads,                                        \
      const T3* moment_1,                                     \
      const T3* moment_2,                                     \
      T3 alpha,                                               \
      T3 beta,                                                \
      T1 lambda,                                              \
      T3 epsilon,                                             \
      T2* weights_out,                                        \
      T3* moment_1_out,                                       \
      T3* moment_2_out,                                       \
      size_t count);

SPECIALIZED_IMPL_LambComputeDirectionImpl(float, float, float)
SPECIALIZED_IMPL_LambComputeDirectionImpl(double, double, double)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, half)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, float)

template <typename T1, typename T2, typename T3, bool update_fp16_weight>
__global__ void _LambUpdate(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T2 threshold,
    const T3* update_direction,
    T2* weights_out,
    half* fp16_weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // The reason to have _Min(...):
  //   The confidence level should not exceed 1 for numerical stability.
  //   The threshold will be used even if r_norm and w_norm are 0 because
  //   NaN > threshold ? NaN : threshold returns threshold.
  // The reason to have *w_norm != 0?:
  //   If a tensor is zero-initialized, its w_norm will be 0 and therefore its
  //   ratio is always 0 without the _Max(...). If a tensor's ratio is always
  //   0, that tensor will never be updated.
  const auto ratio = *w_norm != T2(0.0f)? _Min(*w_norm / *r_norm, threshold) : T2(1.0f);
  // Compute new weight using the saved update direction.
  weights_out[id] = weights[id] - ratio * T2((*eta) * T1(update_direction[id]));

  if (update_fp16_weight) {
    fp16_weights_out[id] = static_cast<half>(weights_out[id]);
  }
}

template <typename T1, typename T2, typename T3>
void LambUpdateImpl(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T2 threshold,
    const T3* update_direction,
    T2* weights_out,
    half* fp16_weights_out,
    size_t count) {
  int blocksPerGrid =
      (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  if (fp16_weights_out != nullptr) {
    _LambUpdate<T1, T2, T3, true><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        eta,
        r_norm,
        w_norm,
        weights,
        threshold,
        update_direction,
        weights_out,
        fp16_weights_out,
        N);
  } else {
    _LambUpdate<T1, T2, T3, false><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
        eta,
        r_norm,
        w_norm,
        weights,
        threshold,
        update_direction,
        weights_out,
        nullptr,
        N);
  }
}

#define SPECIALIZED_IMPL_LambUpdate(T1, T2, T3) \
  template void LambUpdateImpl(                 \
      const T1* eta,                            \
      const T2* r_norm,                         \
      const T2* w_norm,                         \
      const T2* weights,                        \
      const T2 threshold,                       \
      const T3* update_direction,               \
      T2* weights_out,                          \
      half* fp16_weights_out,                   \
      size_t count);

SPECIALIZED_IMPL_LambUpdate(float, float, float)
SPECIALIZED_IMPL_LambUpdate(double, double, double)
SPECIALIZED_IMPL_LambUpdate(half, float, half)
SPECIALIZED_IMPL_LambUpdate(float, float, half)

template <typename T, typename T_GRAD>
__global__ void _AccumulateGradient(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  accumulated_gradient[id] = gradient_buffer[id] + T(gradient[id]);
}

template <typename T, typename T_GRAD>
void AccumulateGradientImpl(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _AccumulateGradient<T, T_GRAD><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      gradient_buffer,
      gradient,
      accumulated_gradient,
      N);
}

#define SPECIALIZED_IMPL_AccumulateGradient(T, T_GRAD) \
  template void AccumulateGradientImpl(                \
      const T* gradient_buffer,                        \
      const T_GRAD* gradient,                          \
      T* accumulated_gradient,                         \
      size_t count);

SPECIALIZED_IMPL_AccumulateGradient(float, float)
SPECIALIZED_IMPL_AccumulateGradient(float, half)

}  // namespace cuda
}  // namespace onnxruntime
