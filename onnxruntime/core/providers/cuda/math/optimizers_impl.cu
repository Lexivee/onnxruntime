#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/cuda/cu_inc/common.cuh"
#include "optimizers.h"
#include "core/providers/cuda/cuda_common.h"


namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void _SGDOptimizer(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  weights_out[id] = weights[id] - ((*eta) * gradients[id]);
}

template <typename T>
void SGDOptimizerImpl(
    const T* eta,
    const T* weights,
    const T* gradients,
    T* weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _SGDOptimizer<T><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      weights,
      gradients,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL__SGDOptimizerImpl(T)      \
template void SGDOptimizerImpl(                    \
    const T* eta,                                  \
    const T* weights,                              \
    const T* gradients,                            \
    T* weights_out,                                \
    size_t count);

SPECIALIZED_IMPL__SGDOptimizerImpl(float)

template <typename T1, typename T2, typename T3, typename T4, typename T_GRAD>
__global__ void _AdamOptimizer(
    const T1* eta,
    const T2* update_count,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    T4 alpha,
    T4 beta,
    T4 lambda,
    T4 epsilon,
    T3* weights_out,
    T4* moment_1_out,
    T4* moment_2_out,
    int64_t* update_count_out,
    half* fp16_weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Regularize gradient.
  const T4 g_regularized = lambda * T4(weights[id]) + T4(grads[id]);

  // A shared constant. 
  const T4 one = T4(1.0f);

  // Update exponentially-averaged historical gradient.
  moment_1_out[id] = \
    alpha * moment_1[id] + (one - alpha) * g_regularized;

  // Update exponentially-averaged historical squared gradient.
  moment_2_out[id] = \
    beta * moment_2[id] + (one  - beta) * g_regularized * g_regularized;

  // Update learning rate - Use the updated eta for the final weight update.
  const T4 count = T4(static_cast<long long>(*update_count));
  const T4 numerator = _Sqrt(one - _Pow(beta, count));
  const T4 denom = one - _Pow(alpha, count);
  const T4 eta_new = T4(*eta) * numerator / denom;

  // Compute the new weight.
  weights_out[id] = weights[id] - \
    T3(eta_new * moment_1_out[id] / (_Sqrt(moment_2_out[id]) + epsilon));
  *update_count_out = (*update_count) + 1;

  if (fp16_weights_out != nullptr) {
    fp16_weights_out[id] = static_cast<half>(weights_out[id]);
  }
}

template <typename T1, typename T2, typename T3, typename T4, typename T_GRAD>
void AdamOptimizerImpl(
    const T1* eta,
    const T2* update_count,
    const T3* weights,
    const T_GRAD* grads,
    const T4* moment_1,
    const T4* moment_2,
    T4 alpha,
    T4 beta,
    T4 lambda,
    T4 epsilon,
    T3* weights_out,
    T4* moment_1_out,
    T4* moment_2_out,
    T2* update_count_out,
    half* fp16_weights_out,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _AdamOptimizer<T1, T2, T3, T4, T_GRAD><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      update_count,
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      weights_out,
      moment_1_out,
      moment_2_out,
      update_count_out,
      fp16_weights_out,
      N);
}

#define SPECIALIZED_AdamOptimizerImpl(T1, T2, T3, T4, T_GRAD) \
template void AdamOptimizerImpl(                              \
    const T1* eta,                                            \
    const T2* update_count,                                   \
    const T3* weights,                                        \
    const T_GRAD* grads,                                      \
    const T4* moment_1,                                       \
    const T4* moment_2,                                       \
    T4 alpha,                                                 \
    T4 beta,                                                  \
    T4 lambda,                                                \
    T4 epsilon,                                               \
    T3* weights_out,                                          \
    T4* moment_1_out,                                         \
    T4* moment_2_out,                                         \
    T2* update_count_out,                                     \
    half* fp16_weights_out,                                   \
    size_t count);

SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, float)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, float)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, float)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, float, half)
SPECIALIZED_AdamOptimizerImpl(half, int64_t, float, half, half)
SPECIALIZED_AdamOptimizerImpl(float, int64_t, float, half, half)

template <typename T1, typename T2, typename T3>
__global__ void _LambComputeDirection(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  const T3 one = T3(1.0);
  const T3 g = T3(grads[id]);

  // Update exponentially-averaged historical gradient
  moment_1_out[id] = alpha * moment_1[id] + \
    (one - alpha) * g;

  // Update exponentially-averaged historical squared gradient
  moment_2_out[id] = beta * moment_2[id] + \
    (one - beta) * g * g;

  // Save regularized update direction to output.
  update_direction[id] = lambda * weights[id] + \
    T1(moment_1_out[id] / (_Sqrt(moment_2_out[id]) + epsilon));
}

template <typename T1, typename T2, typename T3>
void LambComputeDirectionImpl(
    const T1* weights,
    const T2* grads,
    const T3* moment_1,
    const T3* moment_2,
    T3 alpha,
    T3 beta,
    T1 lambda,
    T3 epsilon,
    T2* update_direction,
    T3* moment_1_out,
    T3* moment_2_out,
    size_t count) {
  int blocksPerGrid = \
    (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambComputeDirection<T1, T2, T3>\
    <<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      weights,
      grads,
      moment_1,
      moment_2,
      alpha,
      beta,
      lambda,
      epsilon,
      update_direction,
      moment_1_out,
      moment_2_out,
      N);
}

#define SPECIALIZED_IMPL_LambComputeDirectionImpl(T1, T2, T3) \
template void LambComputeDirectionImpl(                  \
    const T1* weights,                                   \
    const T2* grads,                                     \
    const T3* moment_1,                                  \
    const T3* moment_2,                                  \
    T3 alpha,                                            \
    T3 beta,                                             \
    T1 lambda,                                           \
    T3 epsilon,                                          \
    T2* weights_out,                                     \
    T3* moment_1_out,                                    \
    T3* moment_2_out,                                    \
    size_t count);

SPECIALIZED_IMPL_LambComputeDirectionImpl(float, float, float)
SPECIALIZED_IMPL_LambComputeDirectionImpl(double, double, double)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, half)
SPECIALIZED_IMPL_LambComputeDirectionImpl(float, half, float)

template <typename T1, typename T2>
__global__ void _LambUpdate(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T1* update_direction,
    T2* weights_out,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  // Compute new weight using the saved update direction.
  weights_out[id] = weights[id] - \
    _Sqrt((*w_norm) / (*r_norm)) * T2((*eta) * update_direction[id]);
}

template <typename T1, typename T2>
void LambUpdateImpl(
    const T1* eta,
    const T2* r_norm,
    const T2* w_norm,
    const T2* weights,
    const T1* update_direction,
    T2* weights_out,
    size_t count) {
  int blocksPerGrid = \
    (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _LambUpdate<T1, T2><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      eta,
      r_norm,
      w_norm,
      weights,
      update_direction,
      weights_out,
      N);
}

#define SPECIALIZED_IMPL_LambUpdate(T1, T2) \
template void LambUpdateImpl(               \
    const T1* eta,                          \
    const T2* r_norm,                       \
    const T2* w_norm,                       \
    const T2* weights,                      \
    const T1* update_direction,             \
    T2* weights_out,                        \
    size_t count);

SPECIALIZED_IMPL_LambUpdate(float, float)
SPECIALIZED_IMPL_LambUpdate(double, double)
SPECIALIZED_IMPL_LambUpdate(half, float)

template <typename T, typename T_GRAD>
__global__ void _AccumulateGradient(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    CUDA_LONG N) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, N);
  accumulated_gradient[id] = gradient_buffer[id] + T(gradient[id]);
}

template <typename T, typename T_GRAD>
void AccumulateGradientImpl(
    const T* gradient_buffer,
    const T_GRAD* gradient,
    T* accumulated_gradient,
    size_t count) {
  int blocksPerGrid = (int)(ceil(static_cast<float>(count) / GridDim::maxThreadsPerBlock));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);
  _AccumulateGradient<T, T_GRAD><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0>>>(
      gradient_buffer,
      gradient,
      accumulated_gradient,
      N);
}

#define SPECIALIZED_IMPL_AccumulateGradient(T, T_GRAD)  \
template void AccumulateGradientImpl(                   \
    const T* gradient_buffer,                           \
    const T_GRAD* gradient,                             \
    T* accumulated_gradient,                            \
    size_t count);

SPECIALIZED_IMPL_AccumulateGradient(float, float)
SPECIALIZED_IMPL_AccumulateGradient(float, half)

}  // namespace cuda
}  // namespace onnxruntime
