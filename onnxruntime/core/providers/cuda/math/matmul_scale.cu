#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "matmul_scale.cuh"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "core/mlas/inc/mlas.h"

namespace onnxruntime {
namespace cuda {


template <typename CudaT>
__global__ void ComputeStdDevCoefficientsForScaleKernel(const CudaT* tensor_data, CudaT* d_scale_coef)
{
  static const float scale_coef_power = 1.0f / 3.0f;

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  float val = tensor_data[i];
  float abs_val = fabsf(val);
  d_scale_coef[i] = powf(abs_val, scale_coef_power) * val / abs_val;
}

// h_scale_coef is an array of size num_coef allocated by the caller on the host.
// It will also be subsequently freed by the caller.
void ComputeStdDevCoefficientsForScale(hipStream_t stream, const Tensor* tensor, const int32_t num_coef, MLFloat16* h_scale_coef)
{
  typedef typename ToCudaType<MLFloat16>::MappedType CudaT;
  const CudaT* tensor_data = reinterpret_cast<const CudaT*>(tensor->Data<MLFloat16>());
  int blocksPerGrid = static_cast<int>((num_coef + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock);

  CudaT* d_scale_coef; // Device memory
  hipMalloc(&d_scale_coef, num_coef * sizeof(CudaT)); // Allocate device memory for the kernel output
  ComputeStdDevCoefficientsForScaleKernel<CudaT><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, stream>>>(tensor_data, d_scale_coef);

  hipMemcpyAsync(h_scale_coef, d_scale_coef, num_coef * sizeof(CudaT), hipMemcpyDeviceToHost, stream);  // Copy results back to host
  CUDA_CALL_THROW(hipStreamSynchronize(stream));

  hipFree(d_scale_coef);  // Free device memory
}

}  // namespace cuda
}  // namespace onnxruntime
